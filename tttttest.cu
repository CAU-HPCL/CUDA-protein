#include "hip/hip_runtime.h"
/* include C/C++ header */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

/* include CUDA header */
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>

using namespace cooperative_groups;						// for synchronize between blocks in a single grid

#define _CRT_SECURE_NO_WARINGS


#define WARP_SIZE 32

#define NOT_FOUND -1
#define CODON_SIZE 3

#define RANDOM 0
#define UPPER 1
//#define LOWER 2

#define OBJECTIVE_NUM 3
#define _mCAI 0
#define _mHD 1
#define _MLRCS 2

#define P 0
#define Q 1
#define L 2

//#define FIRST_SOL 1
//#define SECOND_SOL 2


/* -------------------- 20 kinds of amino acids & weights are sorted ascending order -------------------- */
char Amino_abbreviation[20] = { 'A','C','D','E','F','G','H','I','K','L','M','N','P','Q','R','S','T','V','W','Y' };
char Codons[61 * CODON_SIZE + 1] = "GCGGCAGCCGCU\
UGCUGU\
GACGAU\
GAGGAA\
UUUUUC\
GGGGGAGGCGGU\
CACCAU\
AUAAUCAUU\
AAAAAG\
CUCCUGCUUCUAUUAUUG\
AUG\
AAUAAC\
CCGCCCCCUCCA\
CAGCAA\
CGGCGACGCAGGCGUAGA\
UCGAGCAGUUCAUCCUCU\
ACGACAACCACU\
GUAGUGGUCGUU\
UGG\
UAUUAC";
char Codons_num[20] = { 4,2,2,2,2,4,2,3,2,6,1,2,4,2,6,6,4,4,1,2 };
float Codons_weight[61] = { 1854 / 13563.0f, 5296 / 13563.0f, 7223 / 135063.0f, 1.0f,\
1234 / 3052.0f, 1.0f,\
8960 / 12731.0f, 1.0f,\
6172 / 19532.0f,1.0f,\
7773 / 8251.0f, 1.0f,\
1852 / 15694.0f, 2781 / 15694.0f, 3600 / 15694.0f, 1.0f,\
3288 / 4320.0f, 1.0f,\
3172 / 12071.0f, 8251 / 12071.0f,1.0f,\
12845 / 15169.0f, 1.0f,\
1242 / 13329.0f, 2852 / 13329.0f, 3207 / 13329.0f, 4134 / 13329.0f, 8549 / 13329.0f, 1.0f,\
1.0f,\
8613 / 9875.0f,1.0f,\
1064 / 8965.0f, 1656 / 8965.0f, 4575 / 8965.0f, 1.0f,\
3312 / 10987.0f, 1.0f,\
342 / 9784.0f, 489 / 9784.0f, 658 / 9784.0f, 2175 / 9784.0f,3307 / 9784.0f, 1.0f,\
2112 / 10025.0f, 2623 / 10025.0f, 3873 / 10025.0f, 4583 / 10025.0f, 6403 / 10025.0f, 1.0f,\
1938 / 9812.0f, 5037 / 9812.0f,6660 / 9812.0f, 1.0f,\
3249 / 11442.0f, 3700 / 11442.0f, 6911 / 11442.0f, 1.0f,\
1.0f,\
5768 / 7114.0f, 1.0f };
/* ------------------------------ end of definition ------------------------------ */


/* find index of Amino_abbreviation array matching with input amino abbreviation using binary search */
__host__ int FindAminoIndex(char amino_abbreviation)
{
	int low = 0;
	int high = 20 - 1;
	int mid;

	while (low <= high) {
		mid = (low + high) / 2;

		if (Amino_abbreviation[mid] == amino_abbreviation)
			return mid;
		else if (Amino_abbreviation[mid] > amino_abbreviation)
			high = mid - 1;
		else
			low = mid + 1;
	}

	return NOT_FOUND;
}


__device__ char FindNum_C(const char* origin, const char* target, const char num_codons)
{
	char i;

	for (i = 0; i < num_codons; i++)
	{
		if (target[0] == origin[i * CODON_SIZE] && target[1] == origin[i * CODON_SIZE + 1] && target[2] == origin[i * CODON_SIZE + 2]) {
			return i;
		}
	}

	return NOT_FOUND;
}

/* mutate codon upper adaptation or randmom adaptation */
__device__ void mutation(hiprandStateXORWOW* state, const char* codon_info, char* target, char total_num, char origin_pos, const float mprob, const int type)
{
	float cd_prob;
	char new_idx;

	/* 1.0 is included and 0.0 is excluded */
	cd_prob = hiprand_uniform(state);

	switch (type)
	{
	case RANDOM:
		new_idx = (char)(hiprand_uniform(state) * total_num);
		if (cd_prob <= mprob && total_num > 1) {
			while (origin_pos == new_idx || new_idx == total_num) {
				new_idx = (char)(hiprand_uniform(state) * total_num);
			}
			target[0] = codon_info[new_idx * CODON_SIZE];
			target[1] = codon_info[new_idx * CODON_SIZE + 1];
			target[2] = codon_info[new_idx * CODON_SIZE + 2];
		}
		break;

	case UPPER:
		new_idx = (char)(hiprand_uniform(state) * (total_num - 1 - origin_pos));
		if (cd_prob <= mprob && (origin_pos != (total_num - 1))) {
			while (new_idx == (total_num - 1 - origin_pos)) {
				new_idx = (char)(hiprand_uniform(state) * (total_num - 1 - origin_pos));
			}
			target[0] = codon_info[(origin_pos + 1 + new_idx) * CODON_SIZE];
			target[1] = codon_info[(origin_pos + 1 + new_idx) * CODON_SIZE + 1];
			target[2] = codon_info[(origin_pos + 1 + new_idx) * CODON_SIZE + 2];
		}
		break;

	/*case LOWER:
		new_idx = (char)(hiprand_uniform(state) * origin_pos);
		if (cd_prob <= mprob && origin_pos != 0) {
			while (new_idx == origin_pos) {
				new_idx = (char)(hiprand_uniform(state) * origin_pos);
			}
			target[0] = codon_info[new_idx * CODON_SIZE];
			target[1] = codon_info[new_idx * CODON_SIZE + 1];
			target[2] = codon_info[new_idx * CODON_SIZE + 2];
		}
		break;*/
	}

	return;
}


/* hiprand generator state setting */
__global__ void setup_kernel(hiprandStateXORWOW* state, int seed)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	/* Each thread gets same seed, a different sequence number, no offset */
	hiprand_init(seed, id, 0, &state[id]);

	return;
}


__device__ int lock = 0;
__device__ int sorting_idx = 1;


__global__ void mainKernel(hiprandStateXORWOW* state, const char* d_codons, const char* d_codons_num, const float* d_codons_weight, const char* d_amino_seq_idx, const char* d_amino_startpos,
	const int len_amino_seq, const int cds_num, const int cycle, const float mprob, char* d_pop, float* d_objval, char* d_objidx, int* d_lrcsval, 
	int *d_sorted_array, bool *d_check_read, bool *d_check_write)
{
	grid_group g = this_grid();
	
	hiprandStateXORWOW localState;
	int i, j, k, l;
	int idx, seq_idx;
	int num_partition;
	int id;
	char pos;

	int len_cds, len_sol;

	char* ptr_origin_sol, * ptr_target_sol;
	float* ptr_origin_objval, * ptr_target_objval;
	char* ptr_origin_objidx, * ptr_target_objidx;
	int* ptr_origin_lrcsval, * ptr_target_lrcsval;					// P, Q, L

	// for computing MLRCS
	char lrcs_i, lrcs_j;
	int lrcs_p, lrcs_q, lrcs_l, tmp_l;

	//float section_low, section_high, adjust_prob;
	//char direct;
	//int cnt;

	id = blockDim.x * blockIdx.x + threadIdx.x;
	localState = state[id];
	len_cds = len_amino_seq * CODON_SIZE;
	len_sol = len_cds * cds_num;


	/* -------------------- shared memory allocation -------------------- */
	extern __shared__ int smem[];
	/* read only */
	__shared__ char* s_amino_seq_idx;
	__shared__ char* s_amino_startpos;
	__shared__ char* s_codons;
	__shared__ char* s_codons_num;
	__shared__ float* s_codons_weight;
	/* read & write */
	__shared__ char* s_sol1;
	__shared__ char* s_sol2;
	__shared__ char* s_sol1_objidx;
	__shared__ char* s_sol2_objidx;
	__shared__ char* mutation_type;
	__shared__ float* s_obj_compute;										// for computing mCAI & mHD value
	__shared__ float* s_sol1_objval;
	__shared__ float* s_sol2_objval;
	__shared__ int* s_sol1_lrcsval;
	__shared__ int* s_sol2_lrcsval;
	__shared__ int* s_lrcs_tid;

	s_lrcs_tid = smem;
	s_sol1_lrcsval = (int*)&s_lrcs_tid[blockDim.x];							// for finding which thread have LRCS
	s_sol2_lrcsval = (int*)&s_sol1_lrcsval[3];
	s_codons_weight = (float*)&s_sol2_lrcsval[3];
	s_obj_compute = (float*)&s_codons_weight[61];
	s_sol1_objval = (float*)&s_obj_compute[blockDim.x];
	s_sol2_objval = (float*)&s_sol1_objval[OBJECTIVE_NUM];
	s_amino_seq_idx = (char*)&s_sol2_objval[OBJECTIVE_NUM];
	s_amino_startpos = (char*)&s_amino_seq_idx[len_amino_seq];
	s_codons = (char*)&s_amino_startpos[20];
	s_codons_num = (char*)&s_codons[183];
	s_sol1 = (char*)&s_codons_num[20];
	s_sol2 = (char*)&s_sol1[len_sol];
	s_sol1_objidx = (char*)&s_sol2[len_sol];
	s_sol2_objidx = (char*)&s_sol1_objidx[OBJECTIVE_NUM * 2];
	mutation_type = (char*)&s_sol2_objidx[OBJECTIVE_NUM * 2];
	/* -------------------- end of shared memory allocation -------------------- */



	/* read only shared memory variable value setting */
	num_partition = (len_amino_seq % blockDim.x == 0) ? len_amino_seq / blockDim.x : len_amino_seq / blockDim.x + 1;
	for (i = 0; i < num_partition; i++) {			// sequence index
		idx = blockDim.x * i + threadIdx.x;
		if (idx < len_amino_seq)
			s_amino_seq_idx[idx] = d_amino_seq_idx[idx];
	}

	num_partition = 183 / blockDim.x + 1;
	for (i = 0; i < num_partition; i++) {
		idx = blockDim.x * i + threadIdx.x;

		if (idx < 183) {
			s_codons[idx] = d_codons[idx];
		}

		if (idx < 61) {
			s_codons_weight[idx] = d_codons_weight[idx];
		}

		if (idx < 20) {
			s_codons_num[threadIdx.x] = d_codons_num[threadIdx.x];
			s_amino_startpos[threadIdx.x] = d_amino_startpos[threadIdx.x];
		}

	}
	__syncthreads();
	/* ---------- end of initial value setting ---------- */



	/* -------------------- initialize solution -------------------- */
	ptr_origin_sol = s_sol1;
	ptr_origin_objval = s_sol1_objval;
	ptr_origin_objidx = s_sol1_objidx;
	ptr_origin_lrcsval = s_sol1_lrcsval;
	ptr_target_sol = s_sol2;
	ptr_target_objval = s_sol2_objval;
	ptr_target_objidx = s_sol2_objidx;
	ptr_target_lrcsval = s_sol2_lrcsval;

	if (blockIdx.x == gridDim.x - 1)
	{
		num_partition = ((len_amino_seq * cds_num) % blockDim.x == 0) ? (len_amino_seq * cds_num) / blockDim.x : (len_amino_seq * cds_num) / blockDim.x + 1;
		for (i = 0; i < num_partition; i++) {
			idx = blockDim.x * i + threadIdx.x;
			if (idx < len_amino_seq * cds_num) {
				seq_idx = idx % len_amino_seq;

				pos = s_codons_num[s_amino_seq_idx[seq_idx]] - 1;

				j = idx * CODON_SIZE;
				k = (s_amino_startpos[s_amino_seq_idx[seq_idx]] + pos) * CODON_SIZE;

				ptr_origin_sol[j] = s_codons[k];
				ptr_origin_sol[j + 1] = s_codons[k + 1];
				ptr_origin_sol[j + 2] = s_codons[k + 2];
			}
		}
	}
	else {
		num_partition = ((len_amino_seq * cds_num) % blockDim.x == 0) ? (len_amino_seq * cds_num) / blockDim.x : (len_amino_seq * cds_num) / blockDim.x + 1;
		for (i = 0; i < num_partition; i++) {
			idx = blockDim.x * i + threadIdx.x;
			if (idx < len_amino_seq * cds_num) {
				seq_idx = idx % len_amino_seq;

				do {
					pos = (char)(hiprand_uniform(&localState) * s_codons_num[s_amino_seq_idx[seq_idx]]);
				} while (pos == s_codons_num[s_amino_seq_idx[seq_idx]]);

				j = idx * CODON_SIZE;
				k = (s_amino_startpos[s_amino_seq_idx[seq_idx]] + pos) * CODON_SIZE;

				s_sol1[j] = s_codons[k];
				s_sol1[j + 1] = s_codons[k + 1];
				s_sol1[j + 2] = s_codons[k + 2];
			}
		}
	}
	__syncthreads();
	/* -------------------- end of initialize -------------------- */


	/* calculate mCAI */
	num_partition = (len_amino_seq % blockDim.x == 0) ? (len_amino_seq / blockDim.x) : (len_amino_seq / blockDim.x) + 1;
	for (i = 0; i < cds_num; i++) {
		s_obj_compute[threadIdx.x] = 1;

		for (j = 0; j < num_partition; j++) {
			seq_idx = blockDim.x * j + threadIdx.x;
			if (seq_idx < len_amino_seq) {
				pos = FindNum_C(&s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_origin_sol[len_cds * i + seq_idx * CODON_SIZE],
					s_codons_num[s_amino_seq_idx[seq_idx]]);
				s_obj_compute[threadIdx.x] *= (float)pow(s_codons_weight[s_amino_startpos[s_amino_seq_idx[seq_idx]] + pos], 1.0 / len_amino_seq);
			}
		}
		__syncthreads();

		j = blockDim.x / 2;
		while (j != 0) {
			if (threadIdx.x < j) {
				s_obj_compute[threadIdx.x] *= s_obj_compute[threadIdx.x + j];
			}
			__syncthreads();

			j /= 2;
		}

		if (threadIdx.x == 0) {
			if (i == 0) {
				ptr_origin_objval[_mCAI] = s_obj_compute[0];
				ptr_origin_objidx[_mCAI * 2] = i;
			}
			else if (s_obj_compute[0] <= ptr_origin_objval[_mCAI]) {
				ptr_origin_objval[_mCAI] = s_obj_compute[0];
				ptr_origin_objidx[_mCAI * 2] = i;
			}
		}
		__syncthreads();

	}


	/* calculate mHD */
	num_partition = (len_cds % blockDim.x == 0) ? (len_cds / blockDim.x) : (len_cds / blockDim.x) + 1;
	for (i = 0; i < cds_num; i++) {
		for (j = i + 1; j < cds_num; j++) {
			s_obj_compute[threadIdx.x] = 0;

			for (k = 0; k < num_partition; k++) {
				seq_idx = blockDim.x * k + threadIdx.x;

				if (seq_idx < len_cds && (ptr_origin_sol[len_cds * i + seq_idx] != ptr_origin_sol[len_cds * j + seq_idx])) {
					s_obj_compute[threadIdx.x] += 1;
				}
			}
			__syncthreads();

			k = blockDim.x / 2;
			while (k != 0) {
				if (threadIdx.x < k) {
					s_obj_compute[threadIdx.x] += s_obj_compute[threadIdx.x + k];
				}
				__syncthreads();

				k /= 2;
			}

			if (threadIdx.x == 0) {
				if (i == 0 && j == 1) {
					ptr_origin_objval[_mHD] = s_obj_compute[0] / len_cds;
					ptr_origin_objidx[_mHD * 2] = i;
					ptr_origin_objidx[_mHD * 2 + 1] = j;
				}
				else if ((s_obj_compute[0] / len_cds) <= ptr_origin_objval[_mHD]) {
					ptr_origin_objval[_mHD] = s_obj_compute[0] / len_cds;
					ptr_origin_objidx[_mHD * 2] = i;
					ptr_origin_objidx[_mHD * 2 + 1] = j;
				}
			}
			__syncthreads();

		}
	}

	/* calculate MLRCS */
	s_obj_compute[threadIdx.x] = NOT_FOUND;
	lrcs_l = 0;
	for (i = 0; i < cds_num; i++) {
		for (j = i; j < cds_num; j++) {
			idx = threadIdx.x;

			if (i == j)
			{
				while (idx < 2 * len_cds + 1)
				{
					if (idx < len_cds + 1) {
						l = idx + 1;
						seq_idx = len_cds - l;

						for (k = 0; k < l; k++) {
							if (k == 0 || (seq_idx == -1))
								tmp_l = 0;
							else if (ptr_origin_sol[len_cds * i + seq_idx + k] == ptr_origin_sol[len_cds * j + k - 1]) {
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = seq_idx + k + 1 - lrcs_l;
									lrcs_q = k - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}
					}
					else {
						l = 2 * len_cds + 1 - idx;
						seq_idx = len_cds - l;

						for (k = 0; k < l; k++) {
							if (k == 0)
								tmp_l = 0;
							else if (ptr_origin_sol[len_cds * i + k - 1] == ptr_origin_sol[len_cds * j + seq_idx + k])
							{
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = k - lrcs_l;
									lrcs_q = seq_idx + k + 1 - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}

					}

					idx += blockDim.x;
				}
			}
			else
			{
				while (idx < 2 * len_cds + 1)
				{
					if (idx < len_cds + 1) {
						l = idx + 1;
						seq_idx = len_cds - l;
						for (k = 0; k < l; k++) {
							if (k == 0)
								tmp_l = 0;
							else if (ptr_origin_sol[len_cds * i + seq_idx + k] == ptr_origin_sol[len_cds * j + k - 1]) {
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = seq_idx + k + 1 - lrcs_l;
									lrcs_q = k - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}
					}
					else {
						l = 2 * len_cds + 1 - idx;
						seq_idx = len_cds - l;

						for (k = 0; k < l; k++) {
							if (k == 0)
								tmp_l = 0;
							else if (ptr_origin_sol[len_cds * i + k - 1] == ptr_origin_sol[len_cds * j + seq_idx + k])
							{
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = k - lrcs_l;
									lrcs_q = seq_idx + k + 1 - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}

					}

					idx += blockDim.x;
				}
			}

		}
	}
	__syncthreads();

	j = blockDim.x / 2;
	s_lrcs_tid[threadIdx.x] = threadIdx.x;
	__syncthreads();
	while (j != 0)
	{
		if (threadIdx.x < j && (s_obj_compute[threadIdx.x + j] > s_obj_compute[threadIdx.x]))
		{
			s_obj_compute[threadIdx.x] = s_obj_compute[threadIdx.x + j];
			s_lrcs_tid[threadIdx.x] = s_lrcs_tid[threadIdx.x + j];
		}
		__syncthreads();

		j /= 2;
	}

	if (threadIdx.x == s_lrcs_tid[0])
	{
		ptr_origin_lrcsval[L] = lrcs_l;
		ptr_origin_lrcsval[P] = lrcs_p;
		ptr_origin_lrcsval[Q] = lrcs_q;

		ptr_origin_objval[_MLRCS] = (float)lrcs_l / len_cds;
		ptr_origin_objidx[_MLRCS * 2] = lrcs_i;
		ptr_origin_objidx[_MLRCS * 2 + 1] = lrcs_j;
	}
	__syncthreads();




	/* mutate cycle times */
	for (int c = 0; c < cycle; c++)
	{

		/* copy from original solution to target solution */
		num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
		for (i = 0; i < num_partition; i++)
		{
			seq_idx = blockDim.x * i + threadIdx.x;
			if (seq_idx < len_sol)
			{
				ptr_target_sol[seq_idx] = ptr_origin_sol[seq_idx];
			}
		}

		/* select mutatation type */
		if (threadIdx.x == 0) {
			do {
				*mutation_type = (char)(hiprand_uniform(&localState) * 4);
			} while (*mutation_type == 4);
		}
		__syncthreads();


		switch (*mutation_type)
		{
		case 0:			// all random
			num_partition = ((len_amino_seq * cds_num) % blockDim.x == 0) ? (len_amino_seq * cds_num) / blockDim.x : (len_amino_seq * cds_num) / blockDim.x + 1;
			for (i = 0; i < num_partition; i++) {
				idx = blockDim.x * i + threadIdx.x;
				if (idx < len_amino_seq * cds_num) {
					seq_idx = idx % len_amino_seq;

					pos = FindNum_C(&s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_target_sol[idx * CODON_SIZE],
						s_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_target_sol[idx * CODON_SIZE],
						s_codons_num[s_amino_seq_idx[seq_idx]], pos, mprob, RANDOM);
				}
			}
			break;

		case 1:			// mCAI
			num_partition = (len_amino_seq % blockDim.x == 0) ? (len_amino_seq / blockDim.x) : (len_amino_seq / blockDim.x) + 1;
			for (i = 0; i < num_partition; i++) {
				seq_idx = blockDim.x * i + threadIdx.x;
				if (seq_idx < len_amino_seq) {
					pos = FindNum_C(&s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mCAI * 2] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mCAI * 2] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]], pos, mprob, UPPER);
				}
			}
			break;

		case 2:			// mHD
			num_partition = (len_amino_seq % blockDim.x == 0) ? (len_amino_seq / blockDim.x) : (len_amino_seq / blockDim.x) + 1;
			for (i = 0; i < num_partition; i++) {
				seq_idx = blockDim.x * i + threadIdx.x;
				if (seq_idx < len_amino_seq) {
					pos = FindNum_C(&s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]], pos, mprob, RANDOM);

					pos = FindNum_C(&s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2 + 1] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2 + 1] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]], pos, mprob, RANDOM);

				}
			}
			break;

		case 3:
			seq_idx = ptr_origin_lrcsval[P] / CODON_SIZE + threadIdx.x;
			while (seq_idx <= (ptr_origin_lrcsval[P] + ptr_origin_lrcsval[L] - 1) / CODON_SIZE)
			{
				pos = FindNum_C(&s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
					&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]]);
				mutation(&localState, &s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
					&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]], pos, mprob, RANDOM);

				seq_idx += blockDim.x;
			}

			seq_idx = ptr_origin_lrcsval[Q] / CODON_SIZE + threadIdx.x;
			while (seq_idx <= (ptr_origin_lrcsval[Q] + ptr_origin_lrcsval[L] - 1) / CODON_SIZE)
			{
				pos = FindNum_C(&s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
					&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2 + 1] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]]);
				mutation(&localState, &s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
					&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2 + 1] + seq_idx * CODON_SIZE], s_codons_num[s_amino_seq_idx[seq_idx]], pos, mprob, RANDOM);

				seq_idx += blockDim.x;
			}

			break;
		}
		__syncthreads();


		/* calculate mCAI */
		num_partition = (len_amino_seq % blockDim.x == 0) ? (len_amino_seq / blockDim.x) : (len_amino_seq / blockDim.x) + 1;
		for (i = 0; i < cds_num; i++) {
			s_obj_compute[threadIdx.x] = 1;

			for (j = 0; j < num_partition; j++) {
				seq_idx = blockDim.x * j + threadIdx.x;
				if (seq_idx < len_amino_seq) {
					pos = FindNum_C(&s_codons[s_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_target_sol[len_cds * i + seq_idx * CODON_SIZE],
						s_codons_num[s_amino_seq_idx[seq_idx]]);
					s_obj_compute[threadIdx.x] *= (float)pow(s_codons_weight[s_amino_startpos[s_amino_seq_idx[seq_idx]] + pos], 1.0 / len_amino_seq);
				}
			}
			__syncthreads();

			j = blockDim.x / 2;
			while (j != 0) {
				if (threadIdx.x < j) {
					s_obj_compute[threadIdx.x] *= s_obj_compute[threadIdx.x + j];
				}
				__syncthreads();

				j /= 2;
			}

			if (threadIdx.x == 0) {
				if (i == 0) {
					ptr_target_objval[_mCAI] = s_obj_compute[0];
					ptr_target_objidx[_mCAI * 2] = i;
				}
				else if (s_obj_compute[0] <= ptr_target_objval[_mCAI]) {
					ptr_target_objval[_mCAI] = s_obj_compute[0];
					ptr_target_objidx[_mCAI * 2] = i;
				}
			}
			__syncthreads();

		}

		/* calculate mHD */
		num_partition = (len_cds % blockDim.x == 0) ? (len_cds / blockDim.x) : (len_cds / blockDim.x) + 1;
		for (i = 0; i < cds_num; i++) {
			for (j = i + 1; j < cds_num; j++) {
				s_obj_compute[threadIdx.x] = 0;

				for (k = 0; k < num_partition; k++) {
					seq_idx = blockDim.x * k + threadIdx.x;

					if (seq_idx < len_cds && (ptr_target_sol[len_cds * i + seq_idx] != ptr_target_sol[len_cds * j + seq_idx])) {
						s_obj_compute[threadIdx.x] += 1;
					}
				}
				__syncthreads();

				k = blockDim.x / 2;
				while (k != 0) {
					if (threadIdx.x < k) {
						s_obj_compute[threadIdx.x] += s_obj_compute[threadIdx.x + k];
					}
					__syncthreads();

					k /= 2;
				}

				if (threadIdx.x == 0) {
					if (i == 0 && j == 1) {
						ptr_target_objval[_mHD] = s_obj_compute[0] / len_cds;
						ptr_target_objidx[_mHD * 2] = i;
						ptr_target_objidx[_mHD * 2 + 1] = j;
					}
					else if (s_obj_compute[0] / len_cds <= ptr_target_objval[_mHD]) {
						ptr_target_objval[_mHD] = s_obj_compute[0] / len_cds;
						ptr_target_objidx[_mHD * 2] = i;
						ptr_target_objidx[_mHD * 2 + 1] = j;
					}
				}
				__syncthreads();

			}
		}

		/* calculate MLRCS */
		s_obj_compute[threadIdx.x] = NOT_FOUND;
		lrcs_l = 0;
		for (i = 0; i < cds_num; i++) {
			for (j = i; j < cds_num; j++) {
				idx = threadIdx.x;

				if (i == j)
				{
					while (idx < 2 * len_cds + 1)
					{
						if (idx < len_cds + 1) {
							l = idx + 1;
							seq_idx = len_cds - l;

							for (k = 0; k < l; k++) {
								if (k == 0 || (seq_idx == -1))
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + seq_idx + k] == ptr_target_sol[len_cds * j + k - 1]) {
									tmp_l++;
									if (tmp_l >= lrcs_l) {
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = seq_idx + k + 1 - lrcs_l;
										lrcs_q = k - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}
						}
						else {
							l = 2 * len_cds + 1 - idx;
							seq_idx = len_cds - l;

							for (k = 0; k < l; k++) {
								if (k == 0)
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + k - 1] == ptr_target_sol[len_cds * j + seq_idx + k])
								{
									tmp_l++;
									if (tmp_l >= lrcs_l) {
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = k - lrcs_l;
										lrcs_q = seq_idx + k + 1 - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}

						}

						idx += blockDim.x;
					}
				}
				else
				{
					while (idx < 2 * len_cds + 1)
					{
						if (idx < len_cds + 1) {
							l = idx + 1;
							seq_idx = len_cds - l;
							for (k = 0; k < l; k++) {
								if (k == 0)
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + seq_idx + k] == ptr_target_sol[len_cds * j + k - 1]) {
									tmp_l++;
									if (tmp_l >= lrcs_l) {
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = seq_idx + k + 1 - lrcs_l;
										lrcs_q = k - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}
						}
						else {
							l = 2 * len_cds + 1 - idx;
							seq_idx = len_cds - l;

							for (k = 0; k < l; k++) {
								if (k == 0)
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + k - 1] == ptr_target_sol[len_cds * j + seq_idx + k])
								{
									tmp_l++;
									if (tmp_l >= lrcs_l) {
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = k - lrcs_l;
										lrcs_q = seq_idx + k + 1 - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}

						}

						idx += blockDim.x;
					}
				}

			}
		}
		__syncthreads();

		j = blockDim.x / 2;
		s_lrcs_tid[threadIdx.x] = threadIdx.x;
		__syncthreads();
		while (j != 0)
		{
			if (threadIdx.x < j && s_obj_compute[threadIdx.x + j] > s_obj_compute[threadIdx.x])
			{
				s_obj_compute[threadIdx.x] = s_obj_compute[threadIdx.x + j];
				s_lrcs_tid[threadIdx.x] = s_lrcs_tid[threadIdx.x + j];
			}
			__syncthreads();

			j /= 2;
		}

		if (threadIdx.x == s_lrcs_tid[0])
		{
			ptr_target_lrcsval[L] = lrcs_l;
			ptr_target_lrcsval[P] = lrcs_p;
			ptr_target_lrcsval[Q] = lrcs_q;

			ptr_target_objval[_MLRCS] = (float)lrcs_l / len_cds;
			ptr_target_objidx[_MLRCS * 2] = lrcs_i;
			ptr_target_objidx[_MLRCS * 2 + 1] = lrcs_j;
		}
		__syncthreads();



		/* ------------------------------ sorting solutions which are size of 2N ------------------------------ */
		// writing to global memory from shared memory  .... solution, objective value, objective index, lrcs...  
		num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
		for (i = 0; i < num_partition; i++) {
			idx = blockDim.x * i + threadIdx.x;
			if (idx < len_sol) {
				d_pop[blockIdx.x * len_sol + idx] = ptr_origin_sol[idx];
				d_pop[gridDim.x * len_sol + blockIdx.x * len_sol + idx] = ptr_target_sol[idx];
			}
		}
		
		if (threadIdx.x == 0)
		{
			d_objval[blockIdx.x * OBJECTIVE_NUM + _mCAI] = ptr_origin_objval[_mCAI];
			d_objval[blockIdx.x * OBJECTIVE_NUM + _mHD] = ptr_origin_objval[_mHD];
			d_objval[blockIdx.x * OBJECTIVE_NUM + _MLRCS] = ptr_origin_objval[_MLRCS];
			
			d_objval[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM + _mCAI] = ptr_target_objval[_mCAI];
			d_objval[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM + _mHD] = ptr_target_objval[_mHD];
			d_objval[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM + _MLRCS] = ptr_target_objval[_MLRCS];
		
			d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mCAI * 2] = ptr_origin_objidx[_mCAI * 2];
			d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mHD * 2] = ptr_origin_objidx[_mHD * 2];
			d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mHD * 2 + 1] = ptr_origin_objidx[_mHD * 2 + 1];
			d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _MLRCS * 2] = ptr_origin_objidx[_MLRCS * 2];
			d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1] = ptr_origin_objidx[_MLRCS * 2 + 1];

			d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _mCAI * 2] = ptr_target_objidx[_mCAI * 2];
			d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _mHD * 2] = ptr_target_objidx[_mHD * 2];
			d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _mHD * 2 + 1] = ptr_target_objidx[_mHD * 2 + 1];
			d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _MLRCS * 2] = ptr_target_objidx[_MLRCS * 2];
			d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1] = ptr_target_objidx[_MLRCS * 2 + 1];

			d_lrcsval[blockIdx.x * 3 + P] = ptr_origin_lrcsval[P];
			d_lrcsval[blockIdx.x * 3 + Q] = ptr_origin_lrcsval[Q];
			d_lrcsval[blockIdx.x * 3 + L] = ptr_origin_lrcsval[L];

			d_lrcsval[(gridDim.x + blockIdx.x) * 3 + P] = ptr_target_lrcsval[P];
			d_lrcsval[(gridDim.x + blockIdx.x) * 3 + Q] = ptr_target_lrcsval[Q];
			d_lrcsval[(gridDim.x + blockIdx.x) * 3 + L] = ptr_target_lrcsval[L];

			d_check_read[blockIdx.x] = true; 
			d_check_read[gridDim.x + blockIdx.x] = true;
			d_check_write[blockIdx.x] = false;
			d_check_write[gridDim.x + blockIdx.x] = false;
		}
		g.sync();
		// --------------------------------------------------------------------------------------------------------------
		if (blockIdx.x == 0 && threadIdx.x == 0)
			sorting_idx = 0;
		
		num_partition = (gridDim.x * 2 % blockDim.x == 0) ? (gridDim.x * 2 / blockDim.x) : (gridDim.x * 2 / blockDim.x) + 1;
		while (d_check_read[blockIdx.x] || d_check_read[gridDim.x + blockIdx.x]) {
			if (d_check_read[blockIdx.x]) {
				for (i = 0; i < num_partition; i++)
				{
					idx = blockDim.x * i + threadIdx.x;
					if (idx != blockIdx.x && idx < gridDim.x * 2 && d_check_read[idx])
					{
						if (ptr_origin_objval[_mCAI] == d_objval[idx * OBJECTIVE_NUM + _mCAI] &&
							ptr_origin_objval[_mHD] == d_objval[idx * OBJECTIVE_NUM + _mHD] &&
							ptr_origin_objval[_MLRCS] == d_objval[idx * OBJECTIVE_NUM + _MLRCS]
							)
							continue;
						else if (ptr_origin_objval[_mCAI] <= d_objval[idx * OBJECTIVE_NUM + _mCAI] &&
							ptr_origin_objval[_mHD] <= d_objval[idx * OBJECTIVE_NUM + _mHD] &&
							ptr_origin_objval[_MLRCS] >= d_objval[idx * OBJECTIVE_NUM + _MLRCS]
							) 
						{
							d_check_write[blockIdx.x] = true;
							break;
						}
					}
				}
			}

			if (d_check_read[gridDim.x + blockIdx.x]) {
				for (i = 0; i < num_partition; i++)
				{
					idx = blockDim.x * i + threadIdx.x;
					if (idx != gridDim.x + blockIdx.x && idx < gridDim.x * 2 && d_check_read[idx])
					{
						if (ptr_target_objval[_mCAI] == d_objval[idx * OBJECTIVE_NUM + _mCAI] &&
							ptr_target_objval[_mHD] == d_objval[idx * OBJECTIVE_NUM + _mHD] &&
							ptr_target_objval[_MLRCS] == d_objval[idx * OBJECTIVE_NUM + _MLRCS]
							)
							continue;
						else if (ptr_target_objval[_mCAI] <= d_objval[idx * OBJECTIVE_NUM + _mCAI] &&
							ptr_target_objval[_mHD] <= d_objval[idx * OBJECTIVE_NUM + _mHD] &&
							ptr_target_objval[_MLRCS] >= d_objval[idx * OBJECTIVE_NUM + _MLRCS]
							)
						{
							d_check_write[gridDim.x + blockIdx.x] = true;
							break;
						}
					}
				}
			}
			
			g.sync();
			if (threadIdx.x == 0) {
				if (d_check_write[blockIdx.x] == false) {
					while (atomicCAS(&lock, 0, 1) != 0);
					d_sorted_array[sorting_idx++] = blockIdx.x;
					atomicExch(&lock, 0);
				}

				if (d_check_write[gridDim.x + blockIdx.x] == false) {
					while (atomicCAS(&lock, 0, 1) != 0);
					d_sorted_array[sorting_idx++] = gridDim.x + blockIdx.x;
					atomicExch(&lock, 0);
				}
				d_check_read[blockIdx.x] = d_check_write[blockIdx.x];
				d_check_read[gridDim.x + blockIdx.x] = d_check_write[gridDim.x + blockIdx.x];
				d_check_write[blockDim.x] = false;
				d_check_write[gridDim.x + blockDim.x] = false;
			}
			g.sync();
		}
		
		num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
		for (i = 0; i < num_partition; i++) {
			idx = blockDim.x * i + threadIdx.x;
			if (idx < len_sol)
				ptr_origin_sol[idx] = d_pop[d_sorted_array[blockIdx.x] * len_sol + idx];
		}
		if (threadIdx.x == 0) {
			ptr_origin_objval[_mCAI] = d_objval[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM + _mCAI];
			ptr_origin_objval[_mHD] = d_objval[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM + _mHD];
			ptr_origin_objval[_MLRCS] = d_objval[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM + _MLRCS];
			ptr_origin_objidx[_mCAI * 2] = d_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _mCAI * 2];
			ptr_origin_objidx[_mHD * 2] = d_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _mHD * 2];
			ptr_origin_objidx[_mHD * 2 + 1] = d_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _mHD * 2 + 1];
			ptr_origin_objidx[_MLRCS * 2] = d_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _MLRCS * 2];
			ptr_origin_objidx[_MLRCS * 2 + 1] = d_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1];
			ptr_origin_lrcsval[P] = d_lrcsval[d_sorted_array[blockIdx.x] * 3 + P];
			ptr_origin_lrcsval[Q] = d_lrcsval[d_sorted_array[blockIdx.x] * 3 + Q];
			ptr_origin_lrcsval[L] = d_lrcsval[d_sorted_array[blockIdx.x] * 3 + L];
		}
		/* ---------------------------------------- end of sorting ----------------------------------------*/

	}


	/* copy from shared memory to global memory */
	num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
	for (i = 0; i < num_partition; i++) {
		idx = blockDim.x * i + threadIdx.x;
		if (idx < len_sol)
			d_pop[blockIdx.x * len_sol + idx] = ptr_origin_sol[idx];
	}

	if (threadIdx.x == 0)
	{
		d_objval[blockIdx.x * OBJECTIVE_NUM + _mCAI] = ptr_origin_objval[_mCAI];
		d_objval[blockIdx.x * OBJECTIVE_NUM + _mHD] = ptr_origin_objval[_mHD];
		d_objval[blockIdx.x * OBJECTIVE_NUM + _MLRCS] = ptr_origin_objval[_MLRCS];
	}


	return;
}

#define IDEAL_MCAI 1
#define IDEAL_MHD 0.4f
#define IDEAL_MLRCS 0
#define EUCLID(val1, val2, val3) (float)sqrt(pow(IDEAL_MCAI - val1, 2) + pow(IDEAL_MHD - val2, 2) + pow(val3, 2))
/* Minimum distance to optimal objective value(point) */
float MinEuclid(const float* objval, int pop_size)
{
	float res;
	float tmp;

	res = 100;
	for (int i = 0; i < pop_size; i++) {
		tmp = EUCLID(objval[i * OBJECTIVE_NUM + _mCAI], objval[i * OBJECTIVE_NUM + _mHD], objval[i * OBJECTIVE_NUM + _MLRCS]);
		if (tmp < res)
			res = tmp;
	}

	return res;
}



int main()
{
	srand((unsigned int)time(NULL));

	int x;
	float min_dist;
	float mprob;							// mutation probability
	char input_file[32];
	char* amino_seq;						// store amino sequences from input file
	char* h_amino_seq_idx;					// notify index of amino abbreviation array corresponding input amino sequences
	char* h_pop;							// store population (a set of solutions)
	char* h_objidx;
	char* h_amino_startpos;					// notify position of according amino abbreviation index
	int* h_lrcsval;
	int len_amino_seq, len_cds, len_sol;
	int pop_size;
	int cycle;
	int cds_num;							// size of solution equal to number of CDSs(codon sequences) in a solution
	float* h_objval;						// store objective values of population (solution 1, solution 2 .... solution n)
	//float lowest_mcai;						// for divide initial solution section
	//int limit;

	char tmp;
	int i, j, k;
	int idx;
	char buf[256];
	FILE* fp;


	int numBlocks;
	int threadsPerBlock;

	bool* d_check_read, * d_check_write;
	char* d_amino_seq_idx;
	char* d_pop;
	char* d_objidx;
	char* d_amino_startpos;
	char* d_codons;
	char* d_codons_num;
	int* d_lrcsval;
	int* d_sorted_array;
	float* d_objval;
	float* d_codons_weight;
	hiprandStateXORWOW* genState;

	/* for time and mcai section cehck */
	hipEvent_t d_start, d_end;
	float kernel_time;
	hipEventCreate(&d_start);
	hipEventCreate(&d_end);



	/* ---------------------------------------- preprocessing ---------------------------------------- */
	/* input parameter values */
	printf("input file name : "); scanf("%s", input_file);
	printf("input number of cycle : "); scanf("%d", &cycle);					// if number of cycle is zero we can check initial population
	if (cycle < 0) {
		printf("input max cycle value >= 0\n");
		return EXIT_FAILURE;
	}
	printf("input number of solution : "); scanf("%d", &pop_size);
	if (pop_size <= 0) {
		printf("input number of solution > 0\n");
		return EXIT_FAILURE;
	}
	printf("input number of CDSs in a solution : "); scanf("%d", &cds_num);
	if (cds_num <= 1) {
		printf("input number of CDSs > 1\n");
		return EXIT_FAILURE;
	}
	printf("input mutation probability (0 ~ 1 value) : "); scanf("%f", &mprob);
	if (mprob < 0 || mprob > 1) {
		printf("input mutation probability (0 ~ 1 value) : \n");
		return EXIT_FAILURE;
	}
	//printf("input number of limit : "); scanf("%d", &limit);
	printf("input thread per block x value --> number of thread  warp size (32) * x : "); scanf("%d", &x);


	/* read input file (fasta format) */
	fp = fopen(input_file, "r");
	if (fp == NULL) {
		printf("Line : %d Opening input file is failed", __LINE__);
		return EXIT_FAILURE;
	}

	fseek(fp, 0, SEEK_END);
	len_amino_seq = ftell(fp);
	fseek(fp, 0, SEEK_SET);
	fgets(buf, 256, fp);
	len_amino_seq -= ftell(fp);

	amino_seq = (char*)malloc(sizeof(char) * len_amino_seq);

	idx = 0;
	while (!feof(fp)) {
		tmp = fgetc(fp);
		if (tmp != '\n')
			amino_seq[idx++] = tmp;
	}
	amino_seq[idx] = NULL;
	len_amino_seq = idx - 1;
	len_cds = len_amino_seq * CODON_SIZE;
	len_sol = len_cds * cds_num;

	fclose(fp);
	/* end file process */

	h_amino_seq_idx = (char*)malloc(sizeof(char) * len_amino_seq);
	for (i = 0; i < len_amino_seq; i++) {
		idx = FindAminoIndex(amino_seq[i]);
		if (idx == NOT_FOUND) {
			printf("FindAminoIndex function is failed... \n");
			return EXIT_FAILURE;
		}
		h_amino_seq_idx[i] = idx;
	}

	h_amino_startpos = (char*)malloc(sizeof(char) * 20);
	h_amino_startpos[0] = 0;
	for (i = 1; i < 20; i++) {
		h_amino_startpos[i] = h_amino_startpos[i - 1] + Codons_num[i - 1];
	}

	/* caculate the smallest mCAI value */
	//lowest_mcai = 1.f;
	//for (i = 0; i < len_amino_seq; i++) {
	//	lowest_mcai *= (float)pow(Codons_weight[h_amino_startpos[h_amino_seq_idx[i]]], 1.0 / len_amino_seq);
	//}
	/* ---------------------------------------- end of preprocessing ---------------------------------------- */


	threadsPerBlock = WARP_SIZE * x;
	numBlocks = pop_size;

	/* host memory allocation */
	h_pop = (char*)malloc(sizeof(char) * pop_size * len_sol);
	h_objval = (float*)malloc(sizeof(float) * pop_size * OBJECTIVE_NUM);


	/* device memory allocation */
	hipMalloc((void**)&genState, sizeof(hiprandStateXORWOW) * numBlocks * threadsPerBlock);
	hipMalloc((void**)&d_codons, sizeof(Codons));
	hipMalloc((void**)&d_codons_num, sizeof(Codons_num));
	hipMalloc((void**)&d_codons_weight, sizeof(Codons_weight));
	hipMalloc((void**)&d_amino_seq_idx, sizeof(char) * len_amino_seq);
	hipMalloc((void**)&d_amino_startpos, sizeof(char) * 20);
	hipMalloc((void**)&d_pop, sizeof(char) * numBlocks * len_sol * 2);
	hipMalloc((void**)&d_objval, sizeof(float) * numBlocks * OBJECTIVE_NUM * 2);
	hipMalloc((void**)&d_objidx, sizeof(int) * numBlocks * OBJECTIVE_NUM * 2 * 2);
	hipMalloc((void**)&d_lrcsval, sizeof(int) * numBlocks * 3 * 2);
	hipMalloc((void**)&d_sorted_array, sizeof(int) * numBlocks * 2);
	hipMalloc((void**)&d_check_read, sizeof(bool) * numBlocks * 2);
	hipMalloc((void**)&d_check_write, sizeof(bool) * numBlocks * 2);


	/* memory copy host to device */
	hipMemcpy(d_amino_seq_idx, h_amino_seq_idx, sizeof(char) * len_amino_seq, hipMemcpyHostToDevice);
	hipMemcpy(d_amino_startpos, h_amino_startpos, sizeof(char) * 20, hipMemcpyHostToDevice);
	hipMemcpy(d_codons, Codons, sizeof(Codons), hipMemcpyHostToDevice);
	hipMemcpy(d_codons_num, Codons_num, sizeof(Codons_num), hipMemcpyHostToDevice);
	hipMemcpy(d_codons_weight, Codons_weight, sizeof(Codons_weight), hipMemcpyHostToDevice);


	/* optimize kerenl call */
	setup_kernel << <numBlocks, threadsPerBlock >> > (genState, rand());

	hipEventRecord(d_start);
	mainKernel << <numBlocks, threadsPerBlock,
		sizeof(int)* (threadsPerBlock + 3 * 2) + sizeof(float) * (threadsPerBlock + OBJECTIVE_NUM * 2 + 61) +
		sizeof(char) * (len_sol * 2 + len_amino_seq + OBJECTIVE_NUM * 2 * 2 + 183 + 20 + 20 + 1) >> >
		(genState, d_codons, d_codons_num, d_codons_weight, d_amino_seq_idx, d_amino_startpos, len_amino_seq, cds_num, cycle, mprob
			, d_pop, d_objval, d_objidx, d_lrcsval, d_sorted_array, d_check_read, d_check_write);
	hipEventRecord(d_end);
	hipEventSynchronize(d_end);
	hipEventElapsedTime(&kernel_time, d_start, d_end);


	printf("using shared memory size : %d\n", sizeof(int) * (threadsPerBlock + 3 * 2) + sizeof(float) * (threadsPerBlock + OBJECTIVE_NUM * 2 + 61) +
		sizeof(char) * (len_sol * 2 + len_amino_seq + OBJECTIVE_NUM * 2 * 2 + 183 + 20 + 20 + 1));
	printf("\nGPU kerenl cycle time : %f second\n", kernel_time / 1000.f);
	printf("lowest mcai value : %f\n", lowest_mcai);


	/* memory copy device to host */
	hipMemcpy(h_pop, d_pop, sizeof(char) * numBlocks * len_sol, hipMemcpyDeviceToHost);
	hipMemcpy(h_objval, d_objval, sizeof(float) * numBlocks * OBJECTIVE_NUM, hipMemcpyDeviceToHost);


	// print minimum distance to ideal point
	min_dist = MinEuclid(h_objval, pop_size);
	printf("minimum distance to the ideal point : %f\n", min_dist);

	/* print solution */
	//for (i = 0; i < pop_size; i++)
	//{
	//	printf("%d solution\n", i + 1);
	//	for (j = 0; j < cds_num; j++) {
	//		printf("%d cds : ", j + 1);
	//		for (k = 0; k < len_cds; k++) {
	//			printf("%c", h_pop[len_sol * i + len_cds * j + k]);
	//		}
	//		printf("\n");
	//	}
	//	printf("\n");
	//}


	/* print objective value */
	for (i = 0; i < pop_size; i++)
	{
		printf("%d solution\n", i + 1);
		printf("mCAI : %f mHD : %f MLRCS : %f\n", h_objval[i * OBJECTIVE_NUM + _mCAI], h_objval[i * OBJECTIVE_NUM + _mHD], h_objval[i * OBJECTIVE_NUM + _MLRCS]);
	}


	fp = fopen("test.txt", "w");
	/* for computing hypervolume write file */
	for (i = 0; i < pop_size; i++)
	{
		fprintf(fp, "%f %f %f\n", -h_objval[i * OBJECTIVE_NUM + _mCAI], -h_objval[i * OBJECTIVE_NUM + _mHD] / 0.4, h_objval[i * OBJECTIVE_NUM + _MLRCS]);
	}
	fclose(fp);




	/* free deivce memory */
	hipFree(genState);
	hipFree(d_codons);
	hipFree(d_codons_num);
	hipFree(d_codons_weight);
	hipFree(d_amino_seq_idx);
	hipFree(d_amino_startpos);
	hipFree(d_pop);
	hipFree(d_objval);
	hipFree(d_objidx);
	hipFree(d_lrcsval);
	hipFree(d_sorted_array);
	hipFree(d_check_read);
	hipFree(d_check_write);
	hipEventDestroy(d_start);
	hipEventDestroy(d_end);

	/* free host memory */
	free(amino_seq);
	free(h_amino_seq_idx);
	free(h_amino_startpos);
	free(h_pop);
	free(h_objval);


	return EXIT_SUCCESS;
}