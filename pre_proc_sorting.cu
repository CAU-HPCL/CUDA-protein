#include "hip/hip_runtime.h"
/* include C/C++ header */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <chrono>

/* include CUDA header */
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_cooperative_groups.h>

#define _CRT_SECURE_NO_WARINGS

#define CHECK_CUDA(func)                                               \
	{                                                                  \
		hipError_t status = (func);                                   \
		if (status != hipSuccess)                                     \
		{                                                              \
			printf("CUDA API failed at line %d with error: %s (%d)\n", \
				   __LINE__, hipGetErrorString(status), status);      \
			return EXIT_FAILURE;                                       \
		}                                                              \
	}

#define WARP_SIZE 32

#define NOT_FOUND -1

#define RANDOM 0
#define UPPER 1
#define LOWER 2

#define CODON_SIZE 3
#define OBJECTIVE_NUM 3
#define _mCAI 0
#define _mHD 1
#define _MLRCS 2
#define P 0
#define Q 1
#define L 2

#define FIRST_SOL 1
#define SECOND_SOL 2

#define IDEAL_MCAI 1
#define IDEAL_MHD 1
#define IDEAL_MLRCS 0
#define EUCLID(val1, val2, val3) (float)sqrt(pow(IDEAL_MCAI - val1, 2) + pow(IDEAL_MHD - val2, 2) + pow(val3, 2))

/* -------------------- 20 kinds of amino acids & weights are sorted ascending order -------------------- */
char Amino_abbreviation[20] = {'A', 'C', 'D', 'E', 'F', 'G', 'H', 'I', 'K', 'L', 'M', 'N', 'P', 'Q', 'R', 'S', 'T', 'V', 'W', 'Y'};
char Codons[61 * CODON_SIZE + 1] = "GCGGCAGCCGCU\
UGCUGU\
GACGAU\
GAGGAA\
UUUUUC\
GGGGGAGGCGGU\
CACCAU\
AUAAUCAUU\
AAAAAG\
CUCCUGCUUCUAUUAUUG\
AUG\
AAUAAC\
CCGCCCCCUCCA\
CAGCAA\
CGGCGACGCAGGCGUAGA\
UCGAGCAGUUCAUCCUCU\
ACGACAACCACU\
GUAGUGGUCGUU\
UGG\
UAUUAC";
char Codons_num[20] = {4, 2, 2, 2, 2, 4, 2, 3, 2, 6, 1, 2, 4, 2, 6, 6, 4, 4, 1, 2};
float Codons_weight[61] = {1854 / 13563.0f, 5296 / 13563.0f, 7223 / 135063.0f, 1.0f,
						   1234 / 3052.0f, 1.0f,
						   8960 / 12731.0f, 1.0f,
						   6172 / 19532.0f, 1.0f,
						   7773 / 8251.0f, 1.0f,
						   1852 / 15694.0f, 2781 / 15694.0f, 3600 / 15694.0f, 1.0f,
						   3288 / 4320.0f, 1.0f,
						   3172 / 12071.0f, 8251 / 12071.0f, 1.0f,
						   12845 / 15169.0f, 1.0f,
						   1242 / 13329.0f, 2852 / 13329.0f, 3207 / 13329.0f, 4134 / 13329.0f, 8549 / 13329.0f, 1.0f,
						   1.0f,
						   8613 / 9875.0f, 1.0f,
						   1064 / 8965.0f, 1656 / 8965.0f, 4575 / 8965.0f, 1.0f,
						   3312 / 10987.0f, 1.0f,
						   342 / 9784.0f, 489 / 9784.0f, 658 / 9784.0f, 2175 / 9784.0f, 3307 / 9784.0f, 1.0f,
						   2112 / 10025.0f, 2623 / 10025.0f, 3873 / 10025.0f, 4583 / 10025.0f, 6403 / 10025.0f, 1.0f,
						   1938 / 9812.0f, 5037 / 9812.0f, 6660 / 9812.0f, 1.0f,
						   3249 / 11442.0f, 3700 / 11442.0f, 6911 / 11442.0f, 1.0f,
						   1.0f,
						   5768 / 7114.0f, 1.0f};
/* ------------------------------ end of definition ------------------------------ */

/* find index of Amino_abbreviation array matching with input amino abbreviation using binary search */
__host__ int FindAminoIndex(char amino_abbreviation)
{
	int low = 0;
	int high = 20 - 1;
	int mid;

	while (low <= high)
	{
		mid = (low + high) / 2;

		if (Amino_abbreviation[mid] == amino_abbreviation)
			return mid;
		else if (Amino_abbreviation[mid] > amino_abbreviation)
			high = mid - 1;
		else
			low = mid + 1;
	}

	printf("FindAminoIndex function failure\n");
	exit(EXIT_FAILURE);
}

/* Minimum distance to optimal objective value(point) */
__host__ float MinEuclid(const float *objval, int pop_size)
{
	float res;
	float tmp;

	res = EUCLID(objval[_mCAI], objval[_mHD], objval[_MLRCS]);
	for (int i = 0; i < pop_size; i++)
	{
		tmp = EUCLID(objval[i * OBJECTIVE_NUM + _mCAI], objval[i * OBJECTIVE_NUM + _mHD], objval[i * OBJECTIVE_NUM + _MLRCS]);
		if (tmp < res)
			res = tmp;
	}

	return res;
}

__constant__ char c_amino_startpos[20];
__constant__ char c_codons[61 * CODON_SIZE + 1];
__constant__ char c_codons_num[20];
__constant__ int c_len_amino_seq;
__constant__ int c_cds_num;
__constant__ int c_sort_popsize;
__constant__ float c_codons_weight[61];
__constant__ float c_mprob;

__device__ int lock = 0; // for atomic operation
__device__ int front = 0;
__device__ int count = 0;
__device__ int sorting_idx = 0;

__device__ char FindNum_C(const char *origin, const char *target, char num_codons)
{
	for (char i = 0; i < num_codons; i++)
	{
		if (target[0] == origin[i * CODON_SIZE] && target[1] == origin[i * CODON_SIZE + 1] && target[2] == origin[i * CODON_SIZE + 2])
		{
			return i;
		}
	}

	printf("FindNum_C function failure\n");
}

/* mutate codon upper adaptation or randmom adaptation */
__device__ void mutation(hiprandStateXORWOW *state, const char *codon_info, char *target, char total_num, char origin_pos, const float mprob, const int type)
{
	float cd_prob;
	char new_idx;

	/* 1.0 is included and 0.0 is excluded */
	cd_prob = hiprand_uniform(state);

	switch (type)
	{
	case RANDOM:
		new_idx = (char)(hiprand_uniform(state) * total_num);
		if (cd_prob <= mprob && total_num > 1)
		{
			while (origin_pos == new_idx || new_idx == total_num)
			{
				new_idx = (char)(hiprand_uniform(state) * total_num);
			}
			target[0] = codon_info[new_idx * CODON_SIZE];
			target[1] = codon_info[new_idx * CODON_SIZE + 1];
			target[2] = codon_info[new_idx * CODON_SIZE + 2];
		}
		break;

	case UPPER:
		new_idx = (char)(hiprand_uniform(state) * (total_num - 1 - origin_pos));
		if (cd_prob <= mprob && (origin_pos != (total_num - 1)))
		{
			while (new_idx == (total_num - 1 - origin_pos))
			{
				new_idx = (char)(hiprand_uniform(state) * (total_num - 1 - origin_pos));
			}
			target[0] = codon_info[(origin_pos + 1 + new_idx) * CODON_SIZE];
			target[1] = codon_info[(origin_pos + 1 + new_idx) * CODON_SIZE + 1];
			target[2] = codon_info[(origin_pos + 1 + new_idx) * CODON_SIZE + 2];
		}
		break;

		case LOWER:
			new_idx = (char)(hiprand_uniform(state) * origin_pos);
			if (cd_prob <= mprob && origin_pos != 0) {
				while (new_idx == origin_pos) {
					new_idx = (char)(hiprand_uniform(state) * origin_pos);
				}
				target[0] = codon_info[new_idx * CODON_SIZE];
				target[1] = codon_info[new_idx * CODON_SIZE + 1];
				target[2] = codon_info[new_idx * CODON_SIZE + 2];
			}
			break;
	}

	return;
}

__device__ bool ParetoComparison(const float *new_objval, const float *old_objval)
{
	// weak pareto dominance
	if ((new_objval[_mCAI] == old_objval[_mCAI]) &&
		(new_objval[_mHD] == old_objval[_mHD]) &&
		(new_objval[_MLRCS] == old_objval[_MLRCS]))
		return false;
	else if ((new_objval[_mCAI] >= old_objval[_mCAI]) &&
			 (new_objval[_mHD] >= old_objval[_mHD]) &&
			 (new_objval[_MLRCS] <= old_objval[_MLRCS]))
		return true;
	else
		return false;
}

/* hiprand generator state setting */
__global__ void setup_kernel(hiprandStateXORWOW *state, unsigned long long seed)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	/* Each thread gets same seed, a different sequence number, no offset */
	hiprand_init(seed, id, 0, &state[id]);

	return;
}

/*
Afeter complete GenSolution in global memory
d_sorted_array = 0, 1, .. n - 1
and solution, etc...
*/
__global__ void GenSolution(hiprandStateXORWOW* state, const char* d_amino_seq_idx, char* d_pop, float* d_objval, char* d_objidx, int* d_lrcsval, int* d_sorted_array, const int limit, const float lowest_mcai)
{
	hiprandStateXORWOW localState;
	int id;
	char pos;
	int i, j, k, l;
	int idx, seq_idx;
	int num_partition;
	int len_cds, len_sol;
	char lrcs_i, lrcs_j;
	int lrcs_p, lrcs_q, lrcs_l, tmp_l;


	float section_low, section_high, adjust_prob;
	char direct;
	int cnt;


	id = blockDim.x * blockIdx.x + threadIdx.x;
	localState = state[id];
	len_cds = c_len_amino_seq * CODON_SIZE;
	len_sol = len_cds * c_cds_num;

	extern __shared__ int smem[];
	__shared__ int* s_lrcs_tid;
	__shared__ int* s_sol_lrcsval;
	__shared__ float* s_sol_objval;
	__shared__ float* s_obj_compute;
	__shared__ char* s_amino_seq_idx;
	__shared__ char* s_sol;
	__shared__ char* s_sol_objidx;

	s_lrcs_tid = smem;
	s_sol_lrcsval = (int*)&s_lrcs_tid[blockDim.x];
	s_sol_objval = (float*)&s_sol_lrcsval[3];
	s_obj_compute = (float*)&s_sol_objval[OBJECTIVE_NUM];
	s_amino_seq_idx = (char*)&s_obj_compute[blockDim.x];
	s_sol = (char*)&s_amino_seq_idx[c_len_amino_seq];
	s_sol_objidx = (char*)&s_sol[len_sol];


	num_partition = (c_len_amino_seq % blockDim.x == 0) ? c_len_amino_seq / blockDim.x : c_len_amino_seq / blockDim.x + 1;
	for (i = 0; i < num_partition; i++) {
		idx = blockDim.x * i + threadIdx.x;
		if (idx < c_len_amino_seq) {
			s_amino_seq_idx[idx] = d_amino_seq_idx[idx];
		}
	}
	__syncthreads();
	// --------------------------------------------------------------------------------------


	/* -------------------- initialize solution -------------------- */

	if (blockIdx.x == gridDim.x - 1)
	{
		num_partition = ((c_len_amino_seq * c_cds_num) % blockDim.x == 0) ? (c_len_amino_seq * c_cds_num) / blockDim.x : (c_len_amino_seq * c_cds_num) / blockDim.x + 1;
		for (i = 0; i < num_partition; i++) {
			idx = blockDim.x * i + threadIdx.x;
			if (idx < c_len_amino_seq * c_cds_num) {
				seq_idx = idx % c_len_amino_seq;

				pos = c_codons_num[s_amino_seq_idx[seq_idx]] - 1;

				j = idx * CODON_SIZE;
				k = (c_amino_startpos[s_amino_seq_idx[seq_idx]] + pos) * CODON_SIZE;

				s_sol[j] = c_codons[k];
				s_sol[j + 1] = c_codons[k + 1];
				s_sol[j + 2] = c_codons[k + 2];
			}
		}
	}
	else {
		num_partition = ((c_len_amino_seq * c_cds_num) % blockDim.x == 0) ? (c_len_amino_seq * c_cds_num) / blockDim.x : (c_len_amino_seq * c_cds_num) / blockDim.x + 1;
		for (i = 0; i < num_partition; i++) {
			idx = blockDim.x * i + threadIdx.x;
			if (idx < c_len_amino_seq * c_cds_num) {
				seq_idx = idx % c_len_amino_seq;

				do {
					pos = (char)(hiprand_uniform(&localState) * c_codons_num[s_amino_seq_idx[seq_idx]]);
				} while (pos == c_codons_num[s_amino_seq_idx[seq_idx]]);

				j = idx * CODON_SIZE;
				k = (c_amino_startpos[s_amino_seq_idx[seq_idx]] + pos) * CODON_SIZE;

				s_sol[j] = c_codons[k];
				s_sol[j + 1] = c_codons[k + 1];
				s_sol[j + 2] = c_codons[k + 2];
			}
		}
	}
	__syncthreads();


	/* calculate mCAI */
	num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
	for (i = 0; i < c_cds_num; i++) {
		s_obj_compute[threadIdx.x] = 1;

		for (j = 0; j < num_partition; j++) {
			seq_idx = blockDim.x * j + threadIdx.x;
			if (seq_idx < c_len_amino_seq) {
				pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &s_sol[len_cds * i + seq_idx * CODON_SIZE],
					c_codons_num[s_amino_seq_idx[seq_idx]]);
				s_obj_compute[threadIdx.x] *= (float)pow(c_codons_weight[c_amino_startpos[s_amino_seq_idx[seq_idx]] + pos], 1.0 / c_len_amino_seq);
			}
		}
		__syncthreads();

		j = blockDim.x / 2;
		while (true) {
			if (threadIdx.x < j) {
				s_obj_compute[threadIdx.x] *= s_obj_compute[threadIdx.x + j];
			}
			__syncthreads();

			if (j == 1)
				break;

			if ((j % 2 == 1) && (threadIdx.x == 0))
			{
				s_obj_compute[0] *= s_obj_compute[j - 1];
			}
			__syncthreads();

			j /= 2;
		}

		if (threadIdx.x == 0) {
			if (i == 0) {
				s_sol_objval[_mCAI] = s_obj_compute[0];
				s_sol_objidx[_mCAI * 2] = i;
			}
			else if (s_obj_compute[0] <= s_sol_objval[_mCAI]) {
				s_sol_objval[_mCAI] = s_obj_compute[0];
				s_sol_objidx[_mCAI * 2] = i;
			}
		}
		__syncthreads();

	}



	/* ------------------------------ intentional mutation initail solution for adjusting mCAI ------------------------------ */
	section_low = lowest_mcai + (1 - lowest_mcai) / gridDim.x * (blockIdx.x % gridDim.x);
	section_high = lowest_mcai + (1 - lowest_mcai) / gridDim.x * (blockIdx.x % gridDim.x + 1);

	/* muatate */
	adjust_prob = 1.f;
	cnt = 0;
	// mutate direction
	if (s_sol_objval[_mCAI] < section_low)
		direct = UPPER;
	else if (s_sol_objval[_mCAI] > section_high)
		direct = LOWER;
	if (blockIdx.x != gridDim.x - 1) {
		while (cnt < limit && (s_sol_objval[_mCAI] < section_low || s_sol_objval[_mCAI] > section_high))
		{
			if (s_sol_objval[_mCAI] < section_low) {
				if (direct != UPPER) {
					direct = UPPER;
					adjust_prob /= 2;
				}
				num_partition = ((c_len_amino_seq * c_cds_num) % blockDim.x == 0) ? (c_len_amino_seq * c_cds_num) / blockDim.x : (c_len_amino_seq * c_cds_num) / blockDim.x + 1;
				for (i = 0; i < num_partition; i++) {
					idx = blockDim.x * i + threadIdx.x;
					if (idx < c_len_amino_seq * c_cds_num) {
						seq_idx = idx % c_len_amino_seq;

						pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &s_sol[idx * CODON_SIZE],
							c_codons_num[s_amino_seq_idx[seq_idx]]);
						mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &s_sol[idx * CODON_SIZE],
							c_codons_num[s_amino_seq_idx[seq_idx]], pos, adjust_prob, UPPER);
					}
				}
			}
			else {
				if (direct != LOWER) {
					direct = LOWER;
					adjust_prob /= 2;
				}
				num_partition = ((c_len_amino_seq * c_cds_num) % blockDim.x == 0) ? (c_len_amino_seq * c_cds_num) / blockDim.x : (c_len_amino_seq * c_cds_num) / blockDim.x + 1;
				for (i = 0; i < num_partition; i++) {
					idx = blockDim.x * i + threadIdx.x;
					if (idx < c_len_amino_seq * c_cds_num) {
						seq_idx = idx % c_len_amino_seq;

						pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &s_sol[idx * CODON_SIZE],
							c_codons_num[s_amino_seq_idx[seq_idx]]);
						mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &s_sol[idx * CODON_SIZE],
							c_codons_num[s_amino_seq_idx[seq_idx]], pos, adjust_prob, LOWER);
					}
				}
			}

			/* calculate mCAI value */
			num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
			for (i = 0; i < c_cds_num; i++) {
				s_obj_compute[threadIdx.x] = 1;

				for (j = 0; j < num_partition; j++) {
					seq_idx = blockDim.x * j + threadIdx.x;
					if (seq_idx < c_len_amino_seq) {
						pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &s_sol[len_cds * i + seq_idx * CODON_SIZE],
							c_codons_num[s_amino_seq_idx[seq_idx]]);
						s_obj_compute[threadIdx.x] *= (float)pow(c_codons_weight[c_amino_startpos[s_amino_seq_idx[seq_idx]] + pos], 1.0 / c_len_amino_seq);
					}
				}
				__syncthreads();

				j = blockDim.x / 2;
				while (true) {
					if (threadIdx.x < j) {
						s_obj_compute[threadIdx.x] *= s_obj_compute[threadIdx.x + j];
					}
					__syncthreads();

					if (j == 1)
						break;

					if ((j % 2 == 1) && (threadIdx.x == 0))
					{
						s_obj_compute[0] *= s_obj_compute[j - 1];
					}
					__syncthreads();

					j /= 2;
				}

				if (threadIdx.x == 0) {
					if (i == 0) {
						s_sol_objval[_mCAI] = s_obj_compute[0];
						s_sol_objidx[_mCAI * 2] = i;
					}
					else if (s_obj_compute[0] <= s_sol_objval[_mCAI]) {
						s_sol_objval[_mCAI] = s_obj_compute[0];
						s_sol_objidx[_mCAI * 2] = i;
					}
				}
				__syncthreads();

			}
			cnt++;
		}
	}
	/* ------------------------------ end of intational muation ------------------------------ */



	/* calculate mHD */
	num_partition = (len_cds % blockDim.x == 0) ? (len_cds / blockDim.x) : (len_cds / blockDim.x) + 1;
	for (i = 0; i < c_cds_num - 1; i++) {
		for (j = i + 1; j < c_cds_num; j++) {
			s_obj_compute[threadIdx.x] = 0;

			for (k = 0; k < num_partition; k++) {
				seq_idx = blockDim.x * k + threadIdx.x;

				if (seq_idx < len_cds && (s_sol[len_cds * i + seq_idx] != s_sol[len_cds * j + seq_idx])) {
					s_obj_compute[threadIdx.x] += 1;
				}
			}
			__syncthreads();

			k = blockDim.x / 2;
			while (true) {
				if (threadIdx.x < k) {
					s_obj_compute[threadIdx.x] += s_obj_compute[threadIdx.x + k];
				}
				__syncthreads();

				if (k == 1)
					break;

				if ((k % 2 == 1) && (threadIdx.x == 0))
				{
					s_obj_compute[0] += s_obj_compute[k - 1];
				}
				__syncthreads();

				k /= 2;
			}

			if (threadIdx.x == 0) {
				if (i == 0 && j == 1) {
					s_sol_objval[_mHD] = s_obj_compute[0] / len_cds;
					s_sol_objidx[_mHD * 2] = i;
					s_sol_objidx[_mHD * 2 + 1] = j;
				}
				else if ((s_obj_compute[0] / len_cds) <= s_sol_objval[_mHD]) {
					s_sol_objval[_mHD] = s_obj_compute[0] / len_cds;
					s_sol_objidx[_mHD * 2] = i;
					s_sol_objidx[_mHD * 2 + 1] = j;
				}
			}
			__syncthreads();

		}
	}

	/* calculate MLRCS */
	s_obj_compute[threadIdx.x] = NOT_FOUND;
	lrcs_l = 0;
	for (i = 0; i < c_cds_num; i++) {
		for (j = i; j < c_cds_num; j++) {
			idx = threadIdx.x;

			if (i == j)
			{
				while (idx < 2 * len_cds + 1)
				{
					if (idx < len_cds + 1) {
						l = idx + 1;
						seq_idx = len_cds - l;

						for (k = 0; k < l; k++) {
							if (k == 0 || (seq_idx == -1))
								tmp_l = 0;
							else if (s_sol[len_cds * i + seq_idx + k] == s_sol[len_cds * j + k - 1]) {
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = seq_idx + k + 1 - lrcs_l;
									lrcs_q = k - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}
					}
					else {
						l = 2 * len_cds + 1 - idx;
						seq_idx = len_cds - l;

						for (k = 0; k < l; k++) {
							if (k == 0)
								tmp_l = 0;
							else if (s_sol[len_cds * i + k - 1] == s_sol[len_cds * j + seq_idx + k])
							{
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = k - lrcs_l;
									lrcs_q = seq_idx + k + 1 - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}

					}

					idx += blockDim.x;
				}
			}
			else
			{
				while (idx < 2 * len_cds + 1)
				{
					if (idx < len_cds + 1) {
						l = idx + 1;
						seq_idx = len_cds - l;
						for (k = 0; k < l; k++) {
							if (k == 0)
								tmp_l = 0;
							else if (s_sol[len_cds * i + seq_idx + k] == s_sol[len_cds * j + k - 1]) {
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = seq_idx + k + 1 - lrcs_l;
									lrcs_q = k - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}
					}
					else {
						l = 2 * len_cds + 1 - idx;
						seq_idx = len_cds - l;

						for (k = 0; k < l; k++) {
							if (k == 0)
								tmp_l = 0;
							else if (s_sol[len_cds * i + k - 1] == s_sol[len_cds * j + seq_idx + k])
							{
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = k - lrcs_l;
									lrcs_q = seq_idx + k + 1 - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}

					}

					idx += blockDim.x;
				}
			}

		}
	}
	__syncthreads();

	j = blockDim.x / 2;
	s_lrcs_tid[threadIdx.x] = threadIdx.x;
	__syncthreads();
	while (true)
	{
		if (threadIdx.x < j && (s_obj_compute[threadIdx.x + j] > s_obj_compute[threadIdx.x]))
		{
			s_obj_compute[threadIdx.x] = s_obj_compute[threadIdx.x + j];
			s_lrcs_tid[threadIdx.x] = s_lrcs_tid[threadIdx.x + j];
		}
		__syncthreads();

		if (j == 1)
			break;

		if ((j % 2 == 1) && (threadIdx.x == 0))
		{
			if (s_obj_compute[j - 1] > s_obj_compute[0]) {
				s_obj_compute[0] = s_obj_compute[j - 1];
				s_lrcs_tid[0] = s_lrcs_tid[j - 1];
			}
		}
		__syncthreads();

		j /= 2;
	}

	if (threadIdx.x == s_lrcs_tid[0])
	{
		s_sol_lrcsval[L] = lrcs_l;
		s_sol_lrcsval[P] = lrcs_p;
		s_sol_lrcsval[Q] = lrcs_q;

		s_sol_objval[_MLRCS] = (float)lrcs_l / len_cds;
		s_sol_objidx[_MLRCS * 2] = lrcs_i;
		s_sol_objidx[_MLRCS * 2 + 1] = lrcs_j;
	}
	__syncthreads();
	/* -------------------- end of initialize -------------------- */


	/* copy from shared memory to global memory */
	num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
	for (i = 0; i < num_partition; i++) {
		idx = blockDim.x * i + threadIdx.x;
		if (idx < len_sol)
			d_pop[blockIdx.x * len_sol + idx] = s_sol[idx];
	}

	if (threadIdx.x == 0)
	{
		d_objval[blockIdx.x * OBJECTIVE_NUM + _mCAI] = s_sol_objval[_mCAI];
		d_objval[blockIdx.x * OBJECTIVE_NUM + _mHD] = s_sol_objval[_mHD];
		d_objval[blockIdx.x * OBJECTIVE_NUM + _MLRCS] = s_sol_objval[_MLRCS];

		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mCAI * 2] = s_sol_objidx[_mCAI * 2];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mHD * 2] = s_sol_objidx[_mHD * 2];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mHD * 2 + 1] = s_sol_objidx[_mHD * 2 + 1];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _MLRCS * 2] = s_sol_objidx[_MLRCS * 2];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1] = s_sol_objidx[_MLRCS * 2 + 1];

		d_lrcsval[blockIdx.x * 3 + P] = s_sol_lrcsval[P];
		d_lrcsval[blockIdx.x * 3 + Q] = s_sol_lrcsval[Q];
		d_lrcsval[blockIdx.x * 3 + L] = s_sol_lrcsval[L];

		/* write d_sorted_array */
		d_sorted_array[blockIdx.x] = blockIdx.x;
	}

	state[id] = localState;

	return;
}

typedef struct
{
	int sol_idx;
	float corwding_dist;
	float obj_val[OBJECTIVE_NUM];
} Sol;

__device__ void Sol_assign(Sol *s1, Sol *s2)
{
	int i;

	s1->corwding_dist = s2->corwding_dist;
	s1->sol_idx = s2->sol_idx;
	for (i = 0; i < OBJECTIVE_NUM; i++)
	{
		s1->obj_val[i] = s2->obj_val[i];
	}

	return;
}

__device__ void CompUp(Sol *s1, Sol *s2, int idx)
{
	Sol tmp;

	if (s1->obj_val[idx] > s2->obj_val[idx])
	{
		Sol_assign(&tmp, s1);
		Sol_assign(s1, s2);
		Sol_assign(s2, &tmp);
	}
	return;
}

__device__ void CompDownCrowd(Sol *s1, Sol *s2)
{
	Sol tmp;

	if (s1->corwding_dist < s2->corwding_dist)
	{
		Sol_assign(&tmp, s1);
		Sol_assign(s1, s2);
		Sol_assign(s2, &tmp);
	}

	return;
}

/*
Based on sorting methods on NSGA2 paper
If we input number of threads and sorting function into hipOccupancyMaxActiveBlocksPerMultiprocessor(), we get number of blocks available to use
*/
__global__ void FastSortSolution(int *d_sorted_array, bool *F_set, bool *Sp_set, char *d_pop, float *d_objval, char *d_objidx, int *d_lrcsval)
{
	int i, j;
	int sol_idx;
	int sec1, sec2;

	extern __shared__ int smem[];
	__shared__ Sol *s_sol_struct;
	__shared__ int *s_rank_count;
	__shared__ int *s_np;
	__shared__ float *s_objval;

	s_sol_struct = (Sol *)smem;
	s_rank_count = (int *)&s_sol_struct[blockDim.x];
	s_np = (int *)&s_rank_count[blockDim.x];
	s_objval = (float *)&s_np[blockDim.x];

	s_rank_count[threadIdx.x] = 0;
	s_np[threadIdx.x] = 0;

	/* copy objective value from global memory to shared memory */
	s_objval[threadIdx.x * OBJECTIVE_NUM + _mCAI] = d_objval[threadIdx.x * OBJECTIVE_NUM + _mCAI];
	s_objval[threadIdx.x * OBJECTIVE_NUM + _mHD] = d_objval[threadIdx.x * OBJECTIVE_NUM + _mHD];
	s_objval[threadIdx.x * OBJECTIVE_NUM + _MLRCS] = d_objval[threadIdx.x * OBJECTIVE_NUM + _MLRCS];

	if (threadIdx.x == 0)
	{
		front = 0;
		count = 0;
		sorting_idx = 0;
	}
	__syncthreads();

	/* -------------------- 1st front setting -------------------- */
	for (i = 0; i < c_sort_popsize; i++)
	{
		if (threadIdx.x != i)
		{
			if (ParetoComparison(&s_objval[threadIdx.x * OBJECTIVE_NUM], &s_objval[i * OBJECTIVE_NUM]))
				Sp_set[threadIdx.x * c_sort_popsize + i] = true;
			else if (ParetoComparison(&s_objval[i * OBJECTIVE_NUM], &s_objval[threadIdx.x * OBJECTIVE_NUM]))
				s_np[threadIdx.x] += 1;
		}
	}
	if (s_np[threadIdx.x] == 0)
	{
		F_set[threadIdx.x] = true;
		while (atomicCAS(&lock, 0, 1) != 0)
			;
		d_sorted_array[count] = threadIdx.x;
		count += 1;
		s_rank_count[front] += 1;
		atomicExch(&lock, 0);
	}
	__syncthreads();


	sol_idx = 0;
	// crowding distance sort
	if (count > (c_sort_popsize / 2))
	{
		// write solution to shared memory Sol
		if (F_set[threadIdx.x])
		{
			while (atomicCAS(&lock, 0, 1) != 0)
				;
			sol_idx = sorting_idx++;
			atomicExch(&lock, 0);
			s_sol_struct[sol_idx].sol_idx = threadIdx.x;
			s_sol_struct[sol_idx].corwding_dist = 0;
			s_sol_struct[sol_idx].obj_val[_mCAI] = d_objval[threadIdx.x * OBJECTIVE_NUM + _mCAI];
			s_sol_struct[sol_idx].obj_val[_mHD] = d_objval[threadIdx.x * OBJECTIVE_NUM + _mHD] / IDEAL_MHD;
			s_sol_struct[sol_idx].obj_val[_MLRCS] = d_objval[threadIdx.x * OBJECTIVE_NUM + _MLRCS];
		}
		__syncthreads();

		for (i = 0; i < OBJECTIVE_NUM; i++)
		{
			// sorting objective function ascending order
			sec1 = 1;
			while (sec1 < s_rank_count[front])
			{
				if ((threadIdx.x % (sec1 * 2) < sec1) && ((sec1 * 2 * (threadIdx.x / (sec1 * 2) + 1) - threadIdx.x % (sec1 * 2) - 1) < s_rank_count[front]))
					CompUp(&s_sol_struct[threadIdx.x], &s_sol_struct[sec1 * 2 * (threadIdx.x / (sec1 * 2) + 1) - (threadIdx.x % (sec1 * 2)) - 1], i);

				sec2 = sec1 / 2;
				__syncthreads();

				while (sec2 != 0)
				{
					if ((threadIdx.x % (sec2 * 2) < sec2) && (threadIdx.x + sec2 < s_rank_count[front]))
						CompUp(&s_sol_struct[threadIdx.x], &s_sol_struct[threadIdx.x + sec2], i);
					sec2 /= 2;
					__syncthreads();
				}

				sec1 *= 2;
			}
			__syncthreads();

			if (threadIdx.x < s_rank_count[front])
			{
				if (threadIdx.x == 0)
					s_sol_struct[threadIdx.x].corwding_dist = 10000.f;
				else if (threadIdx.x == s_rank_count[front] - 1)
					s_sol_struct[threadIdx.x].corwding_dist = 10000.f;
				else
					s_sol_struct[threadIdx.x].corwding_dist += s_sol_struct[threadIdx.x + 1].obj_val[i] - s_sol_struct[threadIdx.x - 1].obj_val[i];
			}
			__syncthreads();
		}

		// sort crowding distance descending order
		sec1 = 1;
		while (sec1 < s_rank_count[front])
		{
			if ((threadIdx.x % (sec1 * 2)) < sec1 && ((sec1 * 2 * (threadIdx.x / (sec1 * 2) + 1) - threadIdx.x % (sec1 * 2) - 1) < s_rank_count[front]))
				CompDownCrowd(&s_sol_struct[threadIdx.x], &s_sol_struct[sec1 * 2 * (threadIdx.x / (sec1 * 2) + 1) - (threadIdx.x % (sec1 * 2)) - 1]);

			sec2 = sec1 / 2;
			__syncthreads();

			while (sec2 != 0)
			{
				if ((threadIdx.x % (sec2 * 2) < sec2) && (threadIdx.x + sec2 < s_rank_count[front]))
					CompDownCrowd(&s_sol_struct[threadIdx.x], &s_sol_struct[threadIdx.x + sec2]);
				sec2 /= 2;
				__syncthreads();
			}

			sec1 *= 2;
		}
		__syncthreads();

		if (threadIdx.x < s_rank_count[front])
		{
			d_sorted_array[count - s_rank_count[front] + threadIdx.x] = s_sol_struct[threadIdx.x].sol_idx;
		}

		return;
	}


	/* -------------------- non dominated sort  -------------------- */
	if (threadIdx.x == 0)
		front += 1;
	__syncthreads();
	for (i = 0; i < c_sort_popsize - 1; i++)
	{
		for (j = 0; j < c_sort_popsize; j++)
		{
			if (F_set[(front - 1) * c_sort_popsize + j] && Sp_set[j * c_sort_popsize + threadIdx.x])
			{
				s_np[threadIdx.x] -= 1;
				if (s_np[threadIdx.x] == 0)
				{
					F_set[front * c_sort_popsize + threadIdx.x] = true;
					while (atomicCAS(&lock, 0, 1) != 0)
						;
					d_sorted_array[count] = threadIdx.x;
					count += 1;
					s_rank_count[front] += 1; // rank_count 는 체크를 위한 부분으로 나중에 빠지는 것 가능!
					atomicExch(&lock, 0);
				}
			}
		}
		__syncthreads();

		if (count > (c_sort_popsize / 2))
			break;

		if (threadIdx.x == 0)
			front += 1;
		__syncthreads();
	}

	sol_idx = 0;
	// write solution to shared memory Sol
	if (F_set[front * c_sort_popsize + threadIdx.x])
	{
		while (atomicCAS(&lock, 0, 1) != 0)
			;
		sol_idx = sorting_idx++;
		atomicExch(&lock, 0);
		s_sol_struct[sol_idx].sol_idx = threadIdx.x;
		s_sol_struct[sol_idx].corwding_dist = 0;
		s_sol_struct[sol_idx].obj_val[_mCAI] = d_objval[threadIdx.x * OBJECTIVE_NUM + _mCAI];
		s_sol_struct[sol_idx].obj_val[_mHD] = d_objval[threadIdx.x * OBJECTIVE_NUM + _mHD] / IDEAL_MHD;
		s_sol_struct[sol_idx].obj_val[_MLRCS] = d_objval[threadIdx.x * OBJECTIVE_NUM + _MLRCS];
	}
	__syncthreads();

	for (i = 0; i < OBJECTIVE_NUM; i++)
	{
		// sorting objective function ascending order
		sec1 = 1;
		while (sec1 < s_rank_count[front])
		{
			if ((threadIdx.x % (sec1 * 2) < sec1) && ((sec1 * 2 * (threadIdx.x / (sec1 * 2) + 1) - threadIdx.x % (sec1 * 2) - 1) < s_rank_count[front]))
				CompUp(&s_sol_struct[threadIdx.x], &s_sol_struct[sec1 * 2 * (threadIdx.x / (sec1 * 2) + 1) - (threadIdx.x % (sec1 * 2)) - 1], i);

			sec2 = sec1 / 2;
			__syncthreads();
			while (sec2 != 0)
			{
				if ((threadIdx.x % (sec2 * 2) < sec2) && (threadIdx.x + sec2 < s_rank_count[front]))
					CompUp(&s_sol_struct[threadIdx.x], &s_sol_struct[threadIdx.x + sec2], i);
				sec2 /= 2;
				__syncthreads();
			}

			sec1 *= 2;
		}
		__syncthreads();

		if (threadIdx.x < s_rank_count[front])
		{
			if (threadIdx.x == 0)
				s_sol_struct[threadIdx.x].corwding_dist = 10000.f;
			else if (threadIdx.x == s_rank_count[front] - 1)
				s_sol_struct[threadIdx.x].corwding_dist = 10000.f;
			else
				s_sol_struct[threadIdx.x].corwding_dist += s_sol_struct[threadIdx.x + 1].obj_val[i] - s_sol_struct[threadIdx.x - 1].obj_val[i];
		}
		__syncthreads();
	}

	// sort crowding distance descending order
	sec1 = 1;
	while (sec1 < s_rank_count[front])
	{
		if ((threadIdx.x % (sec1 * 2)) < sec1 && ((sec1 * 2 * (threadIdx.x / (sec1 * 2) + 1) - threadIdx.x % (sec1 * 2) - 1) < s_rank_count[front]))
			CompDownCrowd(&s_sol_struct[threadIdx.x], &s_sol_struct[sec1 * 2 * (threadIdx.x / (sec1 * 2) + 1) - (threadIdx.x % (sec1 * 2)) - 1]);
		__syncthreads();

		sec2 = sec1 / 2;
		while (sec2 != 0)
		{
			if ((threadIdx.x % (sec2 * 2) < sec2) && (threadIdx.x + sec2 < s_rank_count[front]))
				CompDownCrowd(&s_sol_struct[threadIdx.x], &s_sol_struct[threadIdx.x + sec2]);
			sec2 /= 2;
			__syncthreads();
		}
		sec1 *= 2;
	}
	__syncthreads();

	if (threadIdx.x < s_rank_count[front])
	{
		d_sorted_array[count - s_rank_count[front] + threadIdx.x] = s_sol_struct[threadIdx.x].sol_idx;
	}

	return;
}

/*
copy solution from global memory based on sorted array
Not update solution to global memory state of sorted Just write solution to global memory
This means In global memory solution is not sorted after this function
If you want soltion is sorted sorted function call and we get sorted array and update solution based on sorted array
*/
__global__ void mainKernel(hiprandStateXORWOW *state, const char *d_amino_seq_idx, char *d_pop, float *d_objval, char *d_objidx, int *d_lrcsval, const int cycle, char *tmp_pop, float *tmp_objval, char *tmp_objidx, int *tmp_lrcsval, int *d_sorted_array)
{
	hiprandStateXORWOW localState;
	int id;
	char pos;
	int i, j, k, l;
	int idx, seq_idx;
	int num_partition;
	int len_cds, len_sol;
	char lrcs_i, lrcs_j;
	int lrcs_p, lrcs_q, lrcs_l, tmp_l;
	char sol_num;

	char *ptr_origin_sol, *ptr_target_sol;
	float *ptr_origin_objval, *ptr_target_objval;
	char *ptr_origin_objidx, *ptr_target_objidx;
	int *ptr_origin_lrcsval, *ptr_target_lrcsval; // P, Q, L

	id = blockDim.x * blockIdx.x + threadIdx.x;
	localState = state[id];
	len_cds = c_len_amino_seq * CODON_SIZE;
	len_sol = len_cds * c_cds_num;

	/* -------------------- shared memory allocation -------------------- */
	extern __shared__ int smem[];
	__shared__ char *s_amino_seq_idx;
	__shared__ char *s_sol1;
	__shared__ char *s_sol2;
	__shared__ char *s_sol1_objidx;
	__shared__ char *s_sol2_objidx;
	__shared__ char *mutation_type;
	__shared__ float *s_obj_compute; // for computing mCAI & mHD value
	__shared__ float *s_sol1_objval;
	__shared__ float *s_sol2_objval;
	__shared__ int *s_sol1_lrcsval;
	__shared__ int *s_sol2_lrcsval;
	__shared__ int *s_lrcs_tid;

	s_lrcs_tid = smem;
	s_sol1_lrcsval = (int *)&s_lrcs_tid[blockDim.x]; // for finding which thread have LRCS
	s_sol2_lrcsval = (int *)&s_sol1_lrcsval[3];
	s_obj_compute = (float *)&s_sol2_lrcsval[3];
	s_sol1_objval = (float *)&s_obj_compute[blockDim.x];
	s_sol2_objval = (float *)&s_sol1_objval[OBJECTIVE_NUM];
	s_amino_seq_idx = (char *)&s_sol2_objval[OBJECTIVE_NUM];
	s_sol1 = (char *)&s_amino_seq_idx[c_len_amino_seq];
	s_sol2 = (char *)&s_sol1[len_sol];
	s_sol1_objidx = (char *)&s_sol2[len_sol];
	s_sol2_objidx = (char *)&s_sol1_objidx[OBJECTIVE_NUM * 2];
	mutation_type = (char *)&s_sol2_objidx[OBJECTIVE_NUM * 2];
	/* -------------------- end of shared memory allocation -------------------- */

	sol_num = FIRST_SOL;
	ptr_origin_sol = s_sol1;
	ptr_origin_objval = s_sol1_objval;
	ptr_origin_objidx = s_sol1_objidx;
	ptr_origin_lrcsval = s_sol1_lrcsval;
	ptr_target_sol = s_sol2;
	ptr_target_objval = s_sol2_objval;
	ptr_target_objidx = s_sol2_objidx;
	ptr_target_lrcsval = s_sol2_lrcsval;

	num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
	for (i = 0; i < num_partition; i++)
	{
		idx = blockDim.x * i + threadIdx.x;
		if (idx < c_len_amino_seq)
		{
			s_amino_seq_idx[idx] = d_amino_seq_idx[idx];
		}
	}
	// -----------------------------------------------------------------------------------------

	/* copy solution from global memory to shared memory based on sorted array index */
	num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
	for (i = 0; i < num_partition; i++)
	{
		idx = blockDim.x * i + threadIdx.x;
		if (idx < len_sol)
			ptr_origin_sol[idx] = tmp_pop[d_sorted_array[blockIdx.x] * len_sol + idx];
	}

	if (threadIdx.x == 0)
	{
		ptr_origin_objval[_mCAI] = tmp_objval[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM + _mCAI];
		ptr_origin_objval[_mHD] = tmp_objval[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM + _mHD];
		ptr_origin_objval[_MLRCS] = tmp_objval[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM + _MLRCS];

		ptr_origin_objidx[_mCAI * 2] = tmp_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _mCAI * 2];
		ptr_origin_objidx[_mHD * 2] = tmp_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _mHD * 2];
		ptr_origin_objidx[_mHD * 2 + 1] = tmp_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _mHD * 2 + 1];
		ptr_origin_objidx[_MLRCS * 2] = tmp_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _MLRCS * 2];
		ptr_origin_objidx[_MLRCS * 2 + 1] = tmp_objidx[d_sorted_array[blockIdx.x] * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1];

		ptr_origin_lrcsval[P] = tmp_lrcsval[d_sorted_array[blockIdx.x] * 3 + P];
		ptr_origin_lrcsval[Q] = tmp_lrcsval[d_sorted_array[blockIdx.x] * 3 + Q];
		ptr_origin_lrcsval[L] = tmp_lrcsval[d_sorted_array[blockIdx.x] * 3 + L];
	}
	__syncthreads();

	/* mutate cycle times */
	for (int c = 0; c < cycle; c++)
	{
		/* copy from original solution to target solution */
		num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
		for (i = 0; i < num_partition; i++)
		{
			seq_idx = blockDim.x * i + threadIdx.x;
			if (seq_idx < len_sol)
			{
				ptr_target_sol[seq_idx] = ptr_origin_sol[seq_idx];
			}
		}

		/* select mutatation type */
		if (threadIdx.x == 0)
		{
			do
			{
				*mutation_type = (char)(hiprand_uniform(&localState) * 4);
			} while (*mutation_type == 4);
		}
		__syncthreads();

		switch (*mutation_type)
		{
		case 0: // all random
			num_partition = ((c_len_amino_seq * c_cds_num) % blockDim.x == 0) ? (c_len_amino_seq * c_cds_num) / blockDim.x : (c_len_amino_seq * c_cds_num) / blockDim.x + 1;
			for (i = 0; i < num_partition; i++)
			{
				idx = blockDim.x * i + threadIdx.x;
				if (idx < c_len_amino_seq * c_cds_num)
				{
					seq_idx = idx % c_len_amino_seq;

					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_target_sol[idx * CODON_SIZE],
									c_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_target_sol[idx * CODON_SIZE],
							 c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);
				}
			}
			break;

		case 1: // mCAI
			num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
			for (i = 0; i < num_partition; i++)
			{
				seq_idx = blockDim.x * i + threadIdx.x;
				if (seq_idx < c_len_amino_seq)
				{
					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
									&ptr_target_sol[len_cds * ptr_origin_objidx[_mCAI * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
							 &ptr_target_sol[len_cds * ptr_origin_objidx[_mCAI * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, UPPER);
				}
			}
			break;

		case 2: // mHD
			num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
			for (i = 0; i < num_partition; i++)
			{
				seq_idx = blockDim.x * i + threadIdx.x;
				if (seq_idx < c_len_amino_seq)
				{
					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
									&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
							 &ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);

					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
									&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2 + 1] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
							 &ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2 + 1] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);
				}
			}
			break;

		case 3:
			seq_idx = ptr_origin_lrcsval[P] / CODON_SIZE + threadIdx.x;
			while (seq_idx <= (ptr_origin_lrcsval[P] + ptr_origin_lrcsval[L] - 1) / CODON_SIZE)
			{
				pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
								&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
				mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						 &ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);

				seq_idx += blockDim.x;
			}

			seq_idx = ptr_origin_lrcsval[Q] / CODON_SIZE + threadIdx.x;
			while (seq_idx <= (ptr_origin_lrcsval[Q] + ptr_origin_lrcsval[L] - 1) / CODON_SIZE)
			{
				pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
								&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2 + 1] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
				mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						 &ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2 + 1] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);

				seq_idx += blockDim.x;
			}

			break;
		}
		__syncthreads();

		/* calculate mCAI */
		num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
		for (i = 0; i < c_cds_num; i++)
		{
			s_obj_compute[threadIdx.x] = 1;

			for (j = 0; j < num_partition; j++)
			{
				seq_idx = blockDim.x * j + threadIdx.x;
				if (seq_idx < c_len_amino_seq)
				{
					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_target_sol[len_cds * i + seq_idx * CODON_SIZE],
									c_codons_num[s_amino_seq_idx[seq_idx]]);
					s_obj_compute[threadIdx.x] *= (float)pow(c_codons_weight[c_amino_startpos[s_amino_seq_idx[seq_idx]] + pos], 1.0 / c_len_amino_seq);
				}
			}
			__syncthreads();

			j = blockDim.x / 2;
			while (true)
			{
				if (threadIdx.x < j)
				{
					s_obj_compute[threadIdx.x] *= s_obj_compute[threadIdx.x + j];
				}
				__syncthreads();

				if (j == 1)
					break;

				if ((j % 2 == 1) && (threadIdx.x == 0))
				{
					s_obj_compute[0] *= s_obj_compute[j - 1];
				}
				__syncthreads();

				j /= 2;
			}

			if (threadIdx.x == 0)
			{
				if (i == 0)
				{
					ptr_target_objval[_mCAI] = s_obj_compute[0];
					ptr_target_objidx[_mCAI * 2] = i;
				}
				else if (s_obj_compute[0] <= ptr_target_objval[_mCAI])
				{
					ptr_target_objval[_mCAI] = s_obj_compute[0];
					ptr_target_objidx[_mCAI * 2] = i;
				}
			}
			__syncthreads();
		}

		/* calculate mHD */
		num_partition = (len_cds % blockDim.x == 0) ? (len_cds / blockDim.x) : (len_cds / blockDim.x) + 1;
		for (i = 0; i < c_cds_num - 1; i++)
		{
			for (j = i + 1; j < c_cds_num; j++)
			{
				s_obj_compute[threadIdx.x] = 0;

				for (k = 0; k < num_partition; k++)
				{
					seq_idx = blockDim.x * k + threadIdx.x;

					if (seq_idx < len_cds && (ptr_target_sol[len_cds * i + seq_idx] != ptr_target_sol[len_cds * j + seq_idx]))
					{
						s_obj_compute[threadIdx.x] += 1;
					}
				}
				__syncthreads();

				k = blockDim.x / 2;
				while (true)
				{
					if (threadIdx.x < k)
					{
						s_obj_compute[threadIdx.x] += s_obj_compute[threadIdx.x + k];
					}
					__syncthreads();

					if (k == 1)
						break;

					if ((k % 2 == 1) && (threadIdx.x == 0))
					{
						s_obj_compute[0] += s_obj_compute[k - 1];
					}
					__syncthreads();

					k /= 2;
				}

				if (threadIdx.x == 0)
				{
					if (i == 0 && j == 1)
					{
						ptr_target_objval[_mHD] = s_obj_compute[0] / len_cds;
						ptr_target_objidx[_mHD * 2] = i;
						ptr_target_objidx[_mHD * 2 + 1] = j;
					}
					else if (s_obj_compute[0] / len_cds <= ptr_target_objval[_mHD])
					{
						ptr_target_objval[_mHD] = s_obj_compute[0] / len_cds;
						ptr_target_objidx[_mHD * 2] = i;
						ptr_target_objidx[_mHD * 2 + 1] = j;
					}
				}
				__syncthreads();
			}
		}

		/* calculate MLRCS */
		s_obj_compute[threadIdx.x] = NOT_FOUND;
		lrcs_l = 0;
		for (i = 0; i < c_cds_num; i++)
		{
			for (j = i; j < c_cds_num; j++)
			{
				idx = threadIdx.x;

				if (i == j)
				{
					while (idx < 2 * len_cds + 1)
					{
						if (idx < len_cds + 1)
						{
							l = idx + 1;
							seq_idx = len_cds - l;

							for (k = 0; k < l; k++)
							{
								if (k == 0 || (seq_idx == -1))
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + seq_idx + k] == ptr_target_sol[len_cds * j + k - 1])
								{
									tmp_l++;
									if (tmp_l >= lrcs_l)
									{
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = seq_idx + k + 1 - lrcs_l;
										lrcs_q = k - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}
						}
						else
						{
							l = 2 * len_cds + 1 - idx;
							seq_idx = len_cds - l;

							for (k = 0; k < l; k++)
							{
								if (k == 0)
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + k - 1] == ptr_target_sol[len_cds * j + seq_idx + k])
								{
									tmp_l++;
									if (tmp_l >= lrcs_l)
									{
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = k - lrcs_l;
										lrcs_q = seq_idx + k + 1 - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}
						}

						idx += blockDim.x;
					}
				}
				else
				{
					while (idx < 2 * len_cds + 1)
					{
						if (idx < len_cds + 1)
						{
							l = idx + 1;
							seq_idx = len_cds - l;
							for (k = 0; k < l; k++)
							{
								if (k == 0)
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + seq_idx + k] == ptr_target_sol[len_cds * j + k - 1])
								{
									tmp_l++;
									if (tmp_l >= lrcs_l)
									{
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = seq_idx + k + 1 - lrcs_l;
										lrcs_q = k - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}
						}
						else
						{
							l = 2 * len_cds + 1 - idx;
							seq_idx = len_cds - l;

							for (k = 0; k < l; k++)
							{
								if (k == 0)
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + k - 1] == ptr_target_sol[len_cds * j + seq_idx + k])
								{
									tmp_l++;
									if (tmp_l >= lrcs_l)
									{
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = k - lrcs_l;
										lrcs_q = seq_idx + k + 1 - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}
						}

						idx += blockDim.x;
					}
				}
			}
		}
		__syncthreads();

		j = blockDim.x / 2;
		s_lrcs_tid[threadIdx.x] = threadIdx.x;
		__syncthreads();
		while (true)
		{
			if (threadIdx.x < j && (s_obj_compute[threadIdx.x + j] > s_obj_compute[threadIdx.x]))
			{
				s_obj_compute[threadIdx.x] = s_obj_compute[threadIdx.x + j];
				s_lrcs_tid[threadIdx.x] = s_lrcs_tid[threadIdx.x + j];
			}
			__syncthreads();

			if (j == 1)
				break;

			if ((j % 2 == 1) && (threadIdx.x == 0))
			{
				if (s_obj_compute[j - 1] > s_obj_compute[0])
				{
					s_obj_compute[0] = s_obj_compute[j - 1];
					s_lrcs_tid[0] = s_lrcs_tid[j - 1];
				}
			}
			__syncthreads();

			j /= 2;
		}

		if (threadIdx.x == s_lrcs_tid[0])
		{
			ptr_target_lrcsval[L] = lrcs_l;
			ptr_target_lrcsval[P] = lrcs_p;
			ptr_target_lrcsval[Q] = lrcs_q;

			ptr_target_objval[_MLRCS] = (float)lrcs_l / len_cds;
			ptr_target_objidx[_MLRCS * 2] = lrcs_i;
			ptr_target_objidx[_MLRCS * 2 + 1] = lrcs_j;
		}
		__syncthreads();

		if (ParetoComparison(ptr_target_objval, ptr_origin_objval))
		{
			if (sol_num == FIRST_SOL)
				sol_num = SECOND_SOL;
			else
				sol_num = FIRST_SOL;
		}

		if (sol_num == FIRST_SOL)
		{
			ptr_origin_sol = s_sol1;
			ptr_origin_objval = s_sol1_objval;
			ptr_origin_objidx = s_sol1_objidx;
			ptr_origin_lrcsval = s_sol1_lrcsval;
			ptr_target_sol = s_sol2;
			ptr_target_objval = s_sol2_objval;
			ptr_target_objidx = s_sol2_objidx;
			ptr_target_lrcsval = s_sol2_lrcsval;
		}
		else
		{
			ptr_origin_sol = s_sol2;
			ptr_origin_objval = s_sol2_objval;
			ptr_origin_objidx = s_sol2_objidx;
			ptr_origin_lrcsval = s_sol2_lrcsval;
			ptr_target_sol = s_sol1;
			ptr_target_objval = s_sol1_objval;
			ptr_target_objidx = s_sol1_objidx;
			ptr_target_lrcsval = s_sol1_lrcsval;
		}
		__syncthreads();
	}

	/* copy from shared memory to global memory */
	num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
	for (i = 0; i < num_partition; i++)
	{
		idx = blockDim.x * i + threadIdx.x;
		if (idx < len_sol)
		{
			d_pop[blockIdx.x * len_sol + idx] = ptr_origin_sol[idx];
			d_pop[(gridDim.x + blockIdx.x) * len_sol + idx] = ptr_target_sol[idx];
		}
	}

	if (threadIdx.x == 0)
	{
		d_objval[blockIdx.x * OBJECTIVE_NUM + _mCAI] = ptr_origin_objval[_mCAI];
		d_objval[blockIdx.x * OBJECTIVE_NUM + _mHD] = ptr_origin_objval[_mHD];
		d_objval[blockIdx.x * OBJECTIVE_NUM + _MLRCS] = ptr_origin_objval[_MLRCS];
		d_objval[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM + _mCAI] = ptr_target_objval[_mCAI];
		d_objval[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM + _mHD] = ptr_target_objval[_mHD];
		d_objval[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM + _MLRCS] = ptr_target_objval[_MLRCS];

		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mCAI * 2] = ptr_origin_objidx[_mCAI * 2];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mHD * 2] = ptr_origin_objidx[_mHD * 2];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mHD * 2 + 1] = ptr_origin_objidx[_mHD * 2 + 1];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _MLRCS * 2] = ptr_origin_objidx[_MLRCS * 2];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1] = ptr_origin_objidx[_MLRCS * 2 + 1];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _mCAI * 2] = ptr_target_objidx[_mCAI * 2];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _mHD * 2] = ptr_target_objidx[_mHD * 2];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _mHD * 2 + 1] = ptr_target_objidx[_mHD * 2 + 1];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _MLRCS * 2] = ptr_target_objidx[_MLRCS * 2];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1] = ptr_target_objidx[_MLRCS * 2 + 1];

		d_lrcsval[blockIdx.x * 3 + P] = ptr_origin_lrcsval[P];
		d_lrcsval[blockIdx.x * 3 + Q] = ptr_origin_lrcsval[Q];
		d_lrcsval[blockIdx.x * 3 + L] = ptr_origin_lrcsval[L];
		d_lrcsval[(gridDim.x + blockIdx.x) * 3 + P] = ptr_target_lrcsval[P];
		d_lrcsval[(gridDim.x + blockIdx.x) * 3 + Q] = ptr_target_lrcsval[Q];
		d_lrcsval[(gridDim.x + blockIdx.x) * 3 + L] = ptr_target_lrcsval[L];
	}

	state[id] = localState; // update state of random number generator

	return;
}

int main()
{
	srand((unsigned int)time(NULL));

	/* To get information of Deivce */
	int dev = 0; // number of device (GPU)
	int maxSharedMemPerBlock;
	int maxSharedMemPerProcessor;
	int totalConstantMem;
	int maxRegisterPerProcessor;
	int maxRegisterPerBlock;
	int totalMultiProcessor;
	hipDeviceProp_t deviceProp;

	CHECK_CUDA(hipGetDeviceProperties(&deviceProp, dev))
	CHECK_CUDA(hipDeviceGetAttribute(&maxSharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, dev))
	CHECK_CUDA(hipDeviceGetAttribute(&maxSharedMemPerProcessor, hipDeviceAttributeMaxRegistersPerMultiprocessor, dev))
	CHECK_CUDA(hipDeviceGetAttribute(&totalConstantMem, hipDeviceAttributeTotalConstantMemory, dev))
	CHECK_CUDA(hipDeviceGetAttribute(&maxRegisterPerProcessor, hipDeviceAttributeMaxRegistersPerMultiprocessor, dev))
	CHECK_CUDA(hipDeviceGetAttribute(&maxRegisterPerBlock, hipDeviceAttributeMaxRegistersPerBlock, dev))
	CHECK_CUDA(hipDeviceGetAttribute(&totalMultiProcessor, hipDeviceAttributeMultiprocessorCount, dev))

	printf("Device #%d:\n", dev);
	printf("Name: %s\n", deviceProp.name);
	printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("Clock rate: %d MHz\n", deviceProp.clockRate / 1000);
	printf("Global memory size: %lu MB\n", deviceProp.totalGlobalMem / (1024 * 1024));
	printf("Max thread dimensions: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("Max grid dimensions: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("Total constant memory: %d bytes\n", totalConstantMem);
	printf("Max threads per SM: %d\n", deviceProp.maxThreadsPerMultiProcessor);
	printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("Maximum shared memory per SM: %d bytes\n", maxSharedMemPerProcessor);
	printf("Maximum shared memory per block: %d bytes\n", maxSharedMemPerBlock);
	printf("Maximum number of registers per SM: %d\n", maxRegisterPerProcessor);
	printf("Maximum number of registers per block: %d\n", maxRegisterPerBlock);
	printf("Total number of SM in device: %d\n", totalMultiProcessor);
	printf("\n");

	char input_file[32];
	char *amino_seq;		// store amino sequences from input file
	char *h_amino_seq_idx;	// notify index of amino abbreviation array corresponding input amino sequences
	char *h_amino_startpos; // notify position of according amino abbreviation index
	char *h_pop;			// store population (a set of solutions)
	float *h_objval;		// store objective values of population (solution 1, solution 2 .... solution n)
	int len_amino_seq, len_cds, len_sol;
	int pop_size;
	int twice_pop;
	int total_cycle;
	int sorting_cycle;
	int cds_num; // size of solution equal to number of CDSs(codon sequences) in a solution
	float mprob; // mutation probability
	float min_dist;

	float lowest_mcai;						// for divide initial solution section
	int limit;

	char tmp;
	int i, j, k;
	int x;
	int idx;
	char buf[256];
	FILE *fp;

	int numBlocks;
	int threadsPerBlock;

	hiprandStateXORWOW *genState;
	hipEvent_t d_start, d_end;
	char *d_amino_seq_idx;
	char *d_pop;
	float *d_objval;
	char *d_objidx;
	int *d_lrcsval;
	int *d_sorted_array;
	bool *d_F_set, *d_Sp_set;
	int *d_np, *d_rank_count;
	Sol *d_sol_struct;

	char *tmp_pop;
	float *tmp_objval;
	char *tmp_objidx;
	int *tmp_lrcsval;

	float kernel_time;
	float total_time = 0;

	/* input parameter values */
	printf("Input file name : ");
	scanf("%s", input_file);
	printf("Input number of cycle : ");
	scanf("%d", &total_cycle);
	if (total_cycle < 0)
	{
		printf("Input max cycle value >= 0\n");
		return EXIT_FAILURE;
	}
	printf("Input number of sorting cycle : ");
	scanf("%d", &sorting_cycle);
	if (sorting_cycle <= 0)
	{
		printf("Input sorting cycle value > 0\n");
		return EXIT_FAILURE;
	}
	printf("Input number of solution : ");
	scanf("%d", &pop_size);
	if (pop_size <= 0)
	{
		printf("Input number of solution > 0\n");
		return EXIT_FAILURE;
	}
	printf("Input number of CDSs in a solution : ");
	scanf("%d", &cds_num);
	if (cds_num <= 1)
	{
		printf("Input number of CDSs > 1\n");
		return EXIT_FAILURE;
	}
	printf("Input mutation probability (0 ~ 1 value) : ");
	scanf("%f", &mprob);
	if (mprob < 0 || mprob > 1)
	{
		printf("Input mutation probability (0 ~ 1 value) : \n");
		return EXIT_FAILURE;
	}
	printf("input number of limit : "); scanf("%d", &limit);
	printf("Input thread per block x value --> number of thread  warp size (32) * x : ");
	scanf("%d", &x);
	printf("For sorting kernel use number of twice_pop threads per block");

	/* read input file (fasta format) */
	fp = fopen(input_file, "r");
	if (fp == NULL)
	{
		printf("Line : %d Opening input file is failed", __LINE__);
		return EXIT_FAILURE;
	}

	numBlocks = pop_size;
	threadsPerBlock = WARP_SIZE * x;
	twice_pop = pop_size * 2;

	fseek(fp, 0, SEEK_END);
	len_amino_seq = ftell(fp);
	fseek(fp, 0, SEEK_SET);
	fgets(buf, 256, fp);
	len_amino_seq -= ftell(fp);

	amino_seq = (char *)malloc(sizeof(char) * len_amino_seq);

	idx = 0;
	while (!feof(fp))
	{
		tmp = fgetc(fp);
		if (tmp != '\n')
			amino_seq[idx++] = tmp;
	}
	amino_seq[idx] = NULL;
	len_amino_seq = idx - 1;
	len_cds = len_amino_seq * CODON_SIZE;
	len_sol = len_cds * cds_num;

	fclose(fp);
	/* end file process */

	h_amino_seq_idx = (char *)malloc(sizeof(char) * len_amino_seq);
	for (i = 0; i < len_amino_seq; i++)
	{
		idx = FindAminoIndex(amino_seq[i]);
		if (idx == NOT_FOUND)
		{
			printf("FindAminoIndex function is failed... \n");
			return EXIT_FAILURE;
		}
		h_amino_seq_idx[i] = idx;
	}

	h_amino_startpos = (char *)malloc(sizeof(char) * 20);
	h_amino_startpos[0] = 0;
	for (i = 1; i < 20; i++)
	{
		h_amino_startpos[i] = h_amino_startpos[i - 1] + Codons_num[i - 1];
	}

/* caculate the smallest mCAI value */
	lowest_mcai = 1.f;
	for (i = 0; i < len_amino_seq; i++) {
		lowest_mcai *= (float)pow(Codons_weight[h_amino_startpos[h_amino_seq_idx[i]]], 1.0 / len_amino_seq);
	}


	/* host memory allocation */
	h_pop = (char *)malloc(sizeof(char) * twice_pop * len_sol);
	h_objval = (float *)malloc(sizeof(float) * twice_pop * OBJECTIVE_NUM);

	/* device memory allocation */
	CHECK_CUDA(hipEventCreate(&d_start))
	CHECK_CUDA(hipEventCreate(&d_end))
	CHECK_CUDA(hipMalloc((void **)&genState, sizeof(hiprandStateXORWOW) * numBlocks * threadsPerBlock))
	CHECK_CUDA(hipMalloc((void **)&d_amino_seq_idx, sizeof(char) * len_amino_seq))
	CHECK_CUDA(hipMalloc((void **)&d_pop, sizeof(char) * twice_pop * len_sol))
	CHECK_CUDA(hipMalloc((void **)&d_objval, sizeof(float) * twice_pop * OBJECTIVE_NUM))
	CHECK_CUDA(hipMalloc((void **)&d_objidx, sizeof(char) * twice_pop * OBJECTIVE_NUM * 2))
	CHECK_CUDA(hipMalloc((void **)&d_lrcsval, sizeof(int) * twice_pop * 3))
	CHECK_CUDA(hipMalloc((void **)&d_sorted_array, sizeof(int) * twice_pop))
	CHECK_CUDA(hipMalloc((void **)&d_F_set, sizeof(bool) * twice_pop * twice_pop))
	CHECK_CUDA(hipMalloc((void **)&d_Sp_set, sizeof(bool) * twice_pop * twice_pop))
	CHECK_CUDA(hipMalloc((void **)&d_rank_count, sizeof(int) * twice_pop))
	CHECK_CUDA(hipMalloc((void **)&d_np, sizeof(int) * twice_pop))
	CHECK_CUDA(hipMalloc((void **)&d_sol_struct, sizeof(Sol) * twice_pop))

	CHECK_CUDA(hipMalloc((void **)&tmp_pop, sizeof(char) * twice_pop * len_sol))
	CHECK_CUDA(hipMalloc((void **)&tmp_objval, sizeof(float) * twice_pop * OBJECTIVE_NUM))
	CHECK_CUDA(hipMalloc((void **)&tmp_objidx, sizeof(char) * twice_pop * OBJECTIVE_NUM * 2))
	CHECK_CUDA(hipMalloc((void **)&tmp_lrcsval, sizeof(int) * twice_pop * 3))

	/* memory copy host to device */
	CHECK_CUDA(hipMemcpy(d_amino_seq_idx, h_amino_seq_idx, sizeof(char) * len_amino_seq, hipMemcpyHostToDevice))
	CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_codons_weight), Codons_weight, sizeof(Codons_weight)))
	CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_amino_startpos), h_amino_startpos, sizeof(char) * 20))
	CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_codons), Codons, sizeof(Codons)))
	CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_codons_num), Codons_num, sizeof(Codons_num)))
	CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_len_amino_seq), &len_amino_seq, sizeof(int)))
	CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_cds_num), &cds_num, sizeof(int)))
	CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_mprob), &mprob, sizeof(float)))
	CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_sort_popsize), &twice_pop, sizeof(int)))

	/* ------------------------------------------------ kerenl call ----------------------------------------------- */
	/* hiprand generator setting */
	CHECK_CUDA(hipEventRecord(d_start))
	setup_kernel<<<numBlocks, threadsPerBlock>>>(genState, (unsigned long long)rand());
	CHECK_CUDA(hipEventRecord(d_end))
	CHECK_CUDA(hipEventSynchronize(d_end))
	CHECK_CUDA(hipEventElapsedTime(&kernel_time, d_start, d_end))
	total_time += kernel_time / 1000.f;

	/* initialize solution */
	CHECK_CUDA(hipEventRecord(d_start))
	GenSolution<<<numBlocks, threadsPerBlock, sizeof(int) * (threadsPerBlock + 3) + sizeof(float) * (threadsPerBlock + OBJECTIVE_NUM) + sizeof(char) * (len_sol + len_amino_seq + 2 * OBJECTIVE_NUM)>>>(genState, d_amino_seq_idx, d_pop, d_objval, d_objidx, d_lrcsval, d_sorted_array, limit, lowest_mcai);
	CHECK_CUDA(hipEventRecord(d_end))
	CHECK_CUDA(hipEventSynchronize(d_end))
	CHECK_CUDA(hipEventElapsedTime(&kernel_time, d_start, d_end))
	total_time += kernel_time / 1000.f;


	void *args[] = {&d_sorted_array, &d_F_set, &d_Sp_set, &d_pop, &d_objval, &d_objidx, &d_lrcsval};
	j = (total_cycle % sorting_cycle == 0) ? total_cycle / sorting_cycle : total_cycle / sorting_cycle + 1;
	CHECK_CUDA(hipEventRecord(d_start))
	for (i = 0; i < j; i++)
	{
		CHECK_CUDA(hipMemcpy(tmp_pop, d_pop, sizeof(char) * len_sol * twice_pop, hipMemcpyDeviceToDevice))
		CHECK_CUDA(hipMemcpy(tmp_objval, d_objval, sizeof(float) * OBJECTIVE_NUM * twice_pop, hipMemcpyDeviceToDevice))
		CHECK_CUDA(hipMemcpy(tmp_objidx, d_objidx, sizeof(char) * OBJECTIVE_NUM * 2 * twice_pop, hipMemcpyDeviceToDevice))
		CHECK_CUDA(hipMemcpy(tmp_lrcsval, d_lrcsval, sizeof(int) * 3 * twice_pop, hipMemcpyDeviceToDevice))
		if (i == j - 1 && (total_cycle % sorting_cycle != 0))
		{
			mainKernel<<<numBlocks, threadsPerBlock, sizeof(int) * (threadsPerBlock + 3 * 2) + sizeof(float) * (threadsPerBlock + OBJECTIVE_NUM * 2) + sizeof(char) * (len_sol * 2 + len_amino_seq + OBJECTIVE_NUM * 2 * 2 + 1)>>>(genState, d_amino_seq_idx, d_pop, d_objval, d_objidx, d_lrcsval, total_cycle % sorting_cycle, tmp_pop, tmp_objval, tmp_objidx, tmp_lrcsval, d_sorted_array);
		}
		else
		{
			mainKernel<<<numBlocks, threadsPerBlock, sizeof(int) * (threadsPerBlock + 3 * 2) + sizeof(float) * (threadsPerBlock + OBJECTIVE_NUM * 2) + sizeof(char) * (len_sol * 2 + len_amino_seq + OBJECTIVE_NUM * 2 * 2 + 1)>>>(genState, d_amino_seq_idx, d_pop, d_objval, d_objidx, d_lrcsval, sorting_cycle, tmp_pop, tmp_objval, tmp_objidx, tmp_lrcsval, d_sorted_array);
		}
		CHECK_CUDA(hipMemset(d_F_set,false,sizeof(bool)*twice_pop * twice_pop))
		CHECK_CUDA(hipMemset(d_Sp_set,false,sizeof(bool)*twice_pop * twice_pop))
		CHECK_CUDA(hipLaunchKernel((void *)FastSortSolution, 1, twice_pop, args, sizeof(Sol) * twice_pop + sizeof(int) * twice_pop * 2 + sizeof(float) * twice_pop * OBJECTIVE_NUM))
	}
	CHECK_CUDA(hipEventRecord(d_end))
	CHECK_CUDA(hipEventSynchronize(d_end))
	CHECK_CUDA(hipEventElapsedTime(&kernel_time, d_start, d_end))
	total_time += kernel_time / 1000.f;

	/* memory copy device to host */
	CHECK_CUDA(hipMemcpy(h_pop, d_pop, sizeof(char) * twice_pop * len_sol, hipMemcpyDeviceToHost))
	CHECK_CUDA(hipMemcpy(h_objval, d_objval, sizeof(float) * twice_pop * OBJECTIVE_NUM, hipMemcpyDeviceToHost))

	// for compute hypervolume & minimum distance out process
	for (i = 0; i < twice_pop; i++)
	{
		h_objval[i * OBJECTIVE_NUM + _mHD] /= 0.4;
	}
	// print minimum distance to ideal point
	min_dist = MinEuclid(h_objval, twice_pop);
	printf("minimum distance to the ideal point : %f\n", min_dist);

	/* print solution */
	// for (i = 0; i < pop_size * 2; i++)
	//{
	//	printf("%d solution\n", i + 1);
	//	for (j = 0; j < cds_num; j++) {
	//		printf("%d cds : ", j + 1);
	//		for (k = 0; k < len_cds; k++) {
	//			printf("%c", h_pop[len_sol * i + len_cds * j + k]);
	//		}
	//		printf("\n");
	//	}
	//	printf("\n");
	// }

	/* print objective value */
	// printf("\n\n");
	// for (i = 0; i < pop_size * 2; i++)
	//{
	//	printf("%d solution\n", i + 1);
	//	printf("mCAI : %f mHD : %f MLRCS : %f\n", h_objval[i * OBJECTIVE_NUM + _mCAI], h_objval[i * OBJECTIVE_NUM + _mHD], h_objval[i * OBJECTIVE_NUM + _MLRCS]);
	//	printf("mCAI idx : %d mHD idx : %d %d MLRCS idx : %d %d\n", h_objidx[i * OBJECTIVE_NUM * 2 + _mCAI * 2],
	//		h_objidx[i * OBJECTIVE_NUM * 2 + _mHD * 2], h_objidx[i * OBJECTIVE_NUM * 2 + _mHD * 2 + 1],
	//		h_objidx[i * OBJECTIVE_NUM * 2 + _MLRCS * 2], h_objidx[i * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1]);
	//	printf("P : %d Q : %d L : %d\n", h_lrcsval[i * 3 + P], h_lrcsval[i * 3 + Q], h_lrcsval[i * 3 + L]);
	// }

	std::chrono::system_clock::time_point start = std::chrono::system_clock::now();
	fp = fopen("test.txt", "w");
	/* for computing hypervolume write file */
	for (i = 0; i < twice_pop; i++)
	{
		fprintf(fp, "%f %f %f\n", -h_objval[i * OBJECTIVE_NUM + _mCAI], -h_objval[i * OBJECTIVE_NUM + _mHD], h_objval[i * OBJECTIVE_NUM + _MLRCS]);
	}
	fclose(fp);
	std::chrono::duration<double> sec = std::chrono::system_clock::now() - start;
	total_time += static_cast<float>(sec.count());
	printf("\n\n total time : %f\n\n", total_time);

	/* free deivce memory */
	CHECK_CUDA(hipEventDestroy(d_start))
	CHECK_CUDA(hipEventDestroy(d_end))
	CHECK_CUDA(hipFree(genState))
	CHECK_CUDA(hipFree(d_amino_seq_idx))
	CHECK_CUDA(hipFree(d_pop))
	CHECK_CUDA(hipFree(d_objval))
	CHECK_CUDA(hipFree(d_objidx))
	CHECK_CUDA(hipFree(d_lrcsval))
	CHECK_CUDA(hipFree(d_sorted_array))
	CHECK_CUDA(hipFree(d_F_set))
	CHECK_CUDA(hipFree(d_Sp_set))
	CHECK_CUDA(hipFree(d_rank_count))
	CHECK_CUDA(hipFree(d_np))
	CHECK_CUDA(hipFree(d_sol_struct))
	CHECK_CUDA(hipFree(tmp_pop))
	CHECK_CUDA(hipFree(tmp_objval))
	CHECK_CUDA(hipFree(tmp_objidx))
	CHECK_CUDA(hipFree(tmp_lrcsval))

	/* free host memory */
	free(amino_seq);
	free(h_amino_seq_idx);
	free(h_amino_startpos);
	free(h_pop);
	free(h_objval);

	return EXIT_SUCCESS;
}