#include "hip/hip_runtime.h"
/* include C/C++ header */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

/* include CUDA header */
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}       

#define _CRT_SECURE_NO_WARINGS

#define WARP_SIZE 32

#define NOT_FOUND -1
#define CODON_SIZE 3

#define RANDOM 0
#define UPPER 1
//#define LOWER 2

#define OBJECTIVE_NUM 3
#define _mCAI 0
#define _mHD 1
#define _MLRCS 2

#define P 0
#define Q 1
#define L 2

#define FIRST_SOL 1
#define SECOND_SOL 2

#define IDEAL_MCAI 1
#define IDEAL_MHD 1
#define IDEAL_MLRCS 0
#define EUCLID(val1, val2, val3) (float)sqrt(pow(IDEAL_MCAI - val1, 2) + pow(IDEAL_MHD - val2, 2) + pow(val3, 2))



/* -------------------- 20 kinds of amino acids & weights are sorted ascending order -------------------- */
char Amino_abbreviation[20] = { 'A','C','D','E','F','G','H','I','K','L','M','N','P','Q','R','S','T','V','W','Y' };
char Codons[61 * CODON_SIZE + 1] = "GCGGCAGCCGCU\
UGCUGU\
GACGAU\
GAGGAA\
UUUUUC\
GGGGGAGGCGGU\
CACCAU\
AUAAUCAUU\
AAAAAG\
CUCCUGCUUCUAUUAUUG\
AUG\
AAUAAC\
CCGCCCCCUCCA\
CAGCAA\
CGGCGACGCAGGCGUAGA\
UCGAGCAGUUCAUCCUCU\
ACGACAACCACU\
GUAGUGGUCGUU\
UGG\
UAUUAC";
char Codons_num[20] = { 4,2,2,2,2,4,2,3,2,6,1,2,4,2,6,6,4,4,1,2 };
float Codons_weight[61] = { 1854 / 13563.0f, 5296 / 13563.0f, 7223 / 135063.0f, 1.0f,\
1234 / 3052.0f, 1.0f,\
8960 / 12731.0f, 1.0f,\
6172 / 19532.0f,1.0f,\
7773 / 8251.0f, 1.0f,\
1852 / 15694.0f, 2781 / 15694.0f, 3600 / 15694.0f, 1.0f,\
3288 / 4320.0f, 1.0f,\
3172 / 12071.0f, 8251 / 12071.0f,1.0f,\
12845 / 15169.0f, 1.0f,\
1242 / 13329.0f, 2852 / 13329.0f, 3207 / 13329.0f, 4134 / 13329.0f, 8549 / 13329.0f, 1.0f,\
1.0f,\
8613 / 9875.0f,1.0f,\
1064 / 8965.0f, 1656 / 8965.0f, 4575 / 8965.0f, 1.0f,\
3312 / 10987.0f, 1.0f,\
342 / 9784.0f, 489 / 9784.0f, 658 / 9784.0f, 2175 / 9784.0f,3307 / 9784.0f, 1.0f,\
2112 / 10025.0f, 2623 / 10025.0f, 3873 / 10025.0f, 4583 / 10025.0f, 6403 / 10025.0f, 1.0f,\
1938 / 9812.0f, 5037 / 9812.0f,6660 / 9812.0f, 1.0f,\
3249 / 11442.0f, 3700 / 11442.0f, 6911 / 11442.0f, 1.0f,\
1.0f,\
5768 / 7114.0f, 1.0f };
/* ------------------------------ end of definition ------------------------------ */


/* find index of Amino_abbreviation array matching with input amino abbreviation using binary search */
__host__ int FindAminoIndex(char amino_abbreviation)
{
	int low = 0;
	int high = 20 - 1;
	int mid;

	while (low <= high) {
		mid = (low + high) / 2;

		if (Amino_abbreviation[mid] == amino_abbreviation)
			return mid;
		else if (Amino_abbreviation[mid] > amino_abbreviation)
			high = mid - 1;
		else
			low = mid + 1;
	}

	return NOT_FOUND;
}

/* Minimum distance to optimal objective value(point) */
__host__ float MinEuclid(const float* objval, int pop_size)
{
	float res;
	float tmp;

	res = 100;
	for (int i = 0; i < pop_size; i++) {
		tmp = EUCLID(objval[i * OBJECTIVE_NUM + _mCAI], objval[i * OBJECTIVE_NUM + _mHD], objval[i * OBJECTIVE_NUM + _MLRCS]);
		if (tmp < res)
			res = tmp;
	}

	return res;
}


__constant__ float c_codons_weight[61];
__constant__ char c_amino_startpos[20];
__constant__ char c_codons[61 * CODON_SIZE + 1];
__constant__ char c_codons_num[20];
__constant__ int c_len_amino_seq;
__constant__ int c_cds_num;
__constant__ float c_mprob;


__device__ char FindNum_C(const char* origin, const char* target, const char num_codons)
{
	char i;

	for (i = 0; i < num_codons; i++)
	{
		if (target[0] == origin[i * CODON_SIZE] && target[1] == origin[i * CODON_SIZE + 1] && target[2] == origin[i * CODON_SIZE + 2]) {
			return i;
		}
	}

	return NOT_FOUND;
}

/* mutate codon upper adaptation or randmom adaptation */
__device__ void mutation(hiprandStateXORWOW* state, const char* codon_info, char* target, char total_num, char origin_pos, const float mprob, const int type)
{
	float cd_prob;
	char new_idx;

	/* 1.0 is included and 0.0 is excluded */
	cd_prob = hiprand_uniform(state);

	switch (type)
	{
	case RANDOM:
		new_idx = (char)(hiprand_uniform(state) * total_num);
		if (cd_prob <= mprob && total_num > 1) {
			while (origin_pos == new_idx || new_idx == total_num) {
				new_idx = (char)(hiprand_uniform(state) * total_num);
			}
			target[0] = codon_info[new_idx * CODON_SIZE];
			target[1] = codon_info[new_idx * CODON_SIZE + 1];
			target[2] = codon_info[new_idx * CODON_SIZE + 2];
		}
		break;

	case UPPER:
		new_idx = (char)(hiprand_uniform(state) * (total_num - 1 - origin_pos));
		if (cd_prob <= mprob && (origin_pos != (total_num - 1))) {
			while (new_idx == (total_num - 1 - origin_pos)) {
				new_idx = (char)(hiprand_uniform(state) * (total_num - 1 - origin_pos));
			}
			target[0] = codon_info[(origin_pos + 1 + new_idx) * CODON_SIZE];
			target[1] = codon_info[(origin_pos + 1 + new_idx) * CODON_SIZE + 1];
			target[2] = codon_info[(origin_pos + 1 + new_idx) * CODON_SIZE + 2];
		}
		break;

	/*case LOWER:
		new_idx = (char)(hiprand_uniform(state) * origin_pos);
		if (cd_prob <= mprob && origin_pos != 0) {
			while (new_idx == origin_pos) {
				new_idx = (char)(hiprand_uniform(state) * origin_pos);
			}
			target[0] = codon_info[new_idx * CODON_SIZE];
			target[1] = codon_info[new_idx * CODON_SIZE + 1];
			target[2] = codon_info[new_idx * CODON_SIZE + 2];
		}
		break;*/
	}


	return;
}

__device__ bool ParetoComparison(const float* new_objval, const float* old_objval)
{
	// weak pareto dominance
	if ((new_objval[_mCAI] == old_objval[_mCAI]) &&
		(new_objval[_mHD] == old_objval[_mHD]) &&
		(new_objval[_MLRCS] == old_objval[_MLRCS]))
		return false;
	else if ((new_objval[_mCAI] >= old_objval[_mCAI]) &&
		(new_objval[_mHD] >= old_objval[_mHD]) &&
		(new_objval[_MLRCS] <= old_objval[_MLRCS]))
		return true;
	else
		return false;
}

/* hiprand generator state setting */
__global__ void setup_kernel(hiprandStateXORWOW* state, int seed)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	/* Each thread gets same seed, a different sequence number, no offset */
	hiprand_init(seed, id, 0, &state[id]);

	return;
}

__global__ void mainKernel(hiprandStateXORWOW* state, const char* d_amino_seq_idx, char* d_pop, float* d_objval, char* d_objidx, int* d_lrcsval, const int cycle)
{
	hiprandStateXORWOW localState;
	int id;
	char pos;
	int i, j, k, l;
	int idx, seq_idx;
	int num_partition;
	int len_cds, len_sol;
	char lrcs_i, lrcs_j;
	int lrcs_p, lrcs_q, lrcs_l, tmp_l;
	char sol_num;

	char* ptr_origin_sol, * ptr_target_sol;
	float* ptr_origin_objval, * ptr_target_objval;
	char* ptr_origin_objidx, * ptr_target_objidx;
	int* ptr_origin_lrcsval, * ptr_target_lrcsval;					// P, Q, L

	id = blockDim.x * blockIdx.x + threadIdx.x;
	localState = state[id];
	len_cds = c_len_amino_seq * CODON_SIZE;
	len_sol = len_cds * c_cds_num;


	/* -------------------- shared memory allocation -------------------- */
	extern __shared__ int smem[];
	__shared__ char* s_amino_seq_idx;
	__shared__ char* s_sol1;
	__shared__ char* s_sol2;
	__shared__ char* s_sol1_objidx;
	__shared__ char* s_sol2_objidx;
	__shared__ char* mutation_type;
	__shared__ float* s_obj_compute;										// for computing mCAI & mHD value
	__shared__ float* s_sol1_objval;
	__shared__ float* s_sol2_objval;
	__shared__ int* s_sol1_lrcsval;
	__shared__ int* s_sol2_lrcsval;
	__shared__ int* s_lrcs_tid;

	s_lrcs_tid = smem;
	s_sol1_lrcsval = (int*)&s_lrcs_tid[blockDim.x];							// for finding which thread have LRCS
	s_sol2_lrcsval = (int*)&s_sol1_lrcsval[3];
	s_obj_compute = (float*)&s_sol2_lrcsval[3];
	s_sol1_objval = (float*)&s_obj_compute[blockDim.x];
	s_sol2_objval = (float*)&s_sol1_objval[OBJECTIVE_NUM];
	s_amino_seq_idx = (char*)&s_sol2_objval[OBJECTIVE_NUM];
	s_sol1 = (char*)&s_amino_seq_idx[c_len_amino_seq];
	s_sol2 = (char*)&s_sol1[len_sol];
	s_sol1_objidx = (char*)&s_sol2[len_sol];
	s_sol2_objidx = (char*)&s_sol1_objidx[OBJECTIVE_NUM * 2];
	mutation_type = (char*)&s_sol2_objidx[OBJECTIVE_NUM * 2];
	/* -------------------- end of shared memory allocation -------------------- */

	sol_num = FIRST_SOL;
	ptr_origin_sol = s_sol1;
	ptr_origin_objval = s_sol1_objval;
	ptr_origin_objidx = s_sol1_objidx;
	ptr_origin_lrcsval = s_sol1_lrcsval;
	ptr_target_sol = s_sol2;
	ptr_target_objval = s_sol2_objval;
	ptr_target_objidx = s_sol2_objidx;
	ptr_target_lrcsval = s_sol2_lrcsval;

	num_partition = (c_len_amino_seq % blockDim.x == 0) ? c_len_amino_seq / blockDim.x : c_len_amino_seq / blockDim.x + 1;
	for (i = 0; i < num_partition; i++) {
		idx = blockDim.x * i + threadIdx.x;
		if (idx < c_len_amino_seq) {
			s_amino_seq_idx[idx] = d_amino_seq_idx[idx];
		}
	}
	// -----------------------------------------------------------------------------------------


	/* -------------------- initialize solution -------------------- */
	if (blockIdx.x == gridDim.x - 1)
	{
		num_partition = ((c_len_amino_seq * c_cds_num) % blockDim.x == 0) ? (c_len_amino_seq * c_cds_num) / blockDim.x : (c_len_amino_seq * c_cds_num) / blockDim.x + 1;
		for (i = 0; i < num_partition; i++) {
			idx = blockDim.x * i + threadIdx.x;
			if (idx < c_len_amino_seq * c_cds_num) {
				seq_idx = idx % c_len_amino_seq;

				pos = c_codons_num[s_amino_seq_idx[seq_idx]] - 1;

				j = idx * CODON_SIZE;
				k = (c_amino_startpos[s_amino_seq_idx[seq_idx]] + pos) * CODON_SIZE;

				ptr_origin_sol[j] = c_codons[k];
				ptr_origin_sol[j + 1] = c_codons[k + 1];
				ptr_origin_sol[j + 2] = c_codons[k + 2];
			}
		}
	}
	else {
		num_partition = ((c_len_amino_seq * c_cds_num) % blockDim.x == 0) ? (c_len_amino_seq * c_cds_num) / blockDim.x : (c_len_amino_seq * c_cds_num) / blockDim.x + 1;
		for (i = 0; i < num_partition; i++) {
			idx = blockDim.x * i + threadIdx.x;
			if (idx < c_len_amino_seq * c_cds_num) {
				seq_idx = idx % c_len_amino_seq;

				do {
					pos = (char)(hiprand_uniform(&localState) * c_codons_num[s_amino_seq_idx[seq_idx]]);
				} while (pos == c_codons_num[s_amino_seq_idx[seq_idx]]);

				j = idx * CODON_SIZE;
				k = (c_amino_startpos[s_amino_seq_idx[seq_idx]] + pos) * CODON_SIZE;

				ptr_origin_sol[j] = c_codons[k];
				ptr_origin_sol[j + 1] = c_codons[k + 1];
				ptr_origin_sol[j + 2] = c_codons[k + 2];
			}
		}
	}
	__syncthreads();
	/* ------------------------------------------ end of initialize --------------------------------------------- */


	/* calculate mCAI */
	num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
	for (i = 0; i < c_cds_num; i++) {
		s_obj_compute[threadIdx.x] = 1;

		for (j = 0; j < num_partition; j++) {
			seq_idx = blockDim.x * j + threadIdx.x;
			if (seq_idx < c_len_amino_seq) {
				pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_origin_sol[len_cds * i + seq_idx * CODON_SIZE],
					c_codons_num[s_amino_seq_idx[seq_idx]]);
				s_obj_compute[threadIdx.x] *= (float)pow(c_codons_weight[c_amino_startpos[s_amino_seq_idx[seq_idx]] + pos], 1.0 / c_len_amino_seq);
			}
		}
		__syncthreads();

		j = blockDim.x / 2;
		while (j != 0) {
			if (threadIdx.x < j) {
				s_obj_compute[threadIdx.x] *= s_obj_compute[threadIdx.x + j];
			}
			__syncthreads();

			j /= 2;
		}

		if (threadIdx.x == 0) {
			if (i == 0) {
				ptr_origin_objval[_mCAI] = s_obj_compute[0];
				ptr_origin_objidx[_mCAI * 2] = i;
			}
			else if (s_obj_compute[0] <= ptr_origin_objval[_mCAI]) {
				ptr_origin_objval[_mCAI] = s_obj_compute[0];
				ptr_origin_objidx[_mCAI * 2] = i;
			}
		}
		__syncthreads();

	}

	/* calculate mHD */
	num_partition = (len_cds % blockDim.x == 0) ? (len_cds / blockDim.x) : (len_cds / blockDim.x) + 1;
	for (i = 0; i < c_cds_num; i++) {
		for (j = i + 1; j < c_cds_num; j++) {
			s_obj_compute[threadIdx.x] = 0;

			for (k = 0; k < num_partition; k++) {
				seq_idx = blockDim.x * k + threadIdx.x;

				if (seq_idx < len_cds && (ptr_origin_sol[len_cds * i + seq_idx] != ptr_origin_sol[len_cds * j + seq_idx])) {
					s_obj_compute[threadIdx.x] += 1;
				}
			}
			__syncthreads();

			k = blockDim.x / 2;
			while (k != 0) {
				if (threadIdx.x < k) {
					s_obj_compute[threadIdx.x] += s_obj_compute[threadIdx.x + k];
				}
				__syncthreads();

				k /= 2;
			}

			if (threadIdx.x == 0) {
				if (i == 0 && j == 1) {
					ptr_origin_objval[_mHD] = s_obj_compute[0] / len_cds;
					ptr_origin_objidx[_mHD * 2] = i;
					ptr_origin_objidx[_mHD * 2 + 1] = j;
				}
				else if ((s_obj_compute[0] / len_cds) <= ptr_origin_objval[_mHD]) {
					ptr_origin_objval[_mHD] = s_obj_compute[0] / len_cds;
					ptr_origin_objidx[_mHD * 2] = i;
					ptr_origin_objidx[_mHD * 2 + 1] = j;
				}
			}
			__syncthreads();

		}
	}

	/* calculate MLRCS */
	s_obj_compute[threadIdx.x] = NOT_FOUND;
	lrcs_l = 0;
	for (i = 0; i < c_cds_num; i++) {
		for (j = i; j < c_cds_num; j++) {
			idx = threadIdx.x;

			if (i == j)
			{
				while (idx < 2 * len_cds + 1)
				{
					if (idx < len_cds + 1) {
						l = idx + 1;
						seq_idx = len_cds - l;

						for (k = 0; k < l; k++) {
							if (k == 0 || (seq_idx == -1))
								tmp_l = 0;
							else if (ptr_origin_sol[len_cds * i + seq_idx + k] == ptr_origin_sol[len_cds * j + k - 1]) {
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = seq_idx + k + 1 - lrcs_l;
									lrcs_q = k - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}
					}
					else {
						l = 2 * len_cds + 1 - idx;
						seq_idx = len_cds - l;

						for (k = 0; k < l; k++) {
							if (k == 0)
								tmp_l = 0;
							else if (ptr_origin_sol[len_cds * i + k - 1] == ptr_origin_sol[len_cds * j + seq_idx + k])
							{
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = k - lrcs_l;
									lrcs_q = seq_idx + k + 1 - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}

					}

					idx += blockDim.x;
				}
			}
			else
			{
				while (idx < 2 * len_cds + 1)
				{
					if (idx < len_cds + 1) {
						l = idx + 1;
						seq_idx = len_cds - l;
						for (k = 0; k < l; k++) {
							if (k == 0)
								tmp_l = 0;
							else if (ptr_origin_sol[len_cds * i + seq_idx + k] == ptr_origin_sol[len_cds * j + k - 1]) {
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = seq_idx + k + 1 - lrcs_l;
									lrcs_q = k - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}
					}
					else {
						l = 2 * len_cds + 1 - idx;
						seq_idx = len_cds - l;

						for (k = 0; k < l; k++) {
							if (k == 0)
								tmp_l = 0;
							else if (ptr_origin_sol[len_cds * i + k - 1] == ptr_origin_sol[len_cds * j + seq_idx + k])
							{
								tmp_l++;
								if (tmp_l >= lrcs_l) {
									lrcs_l = tmp_l;
									s_obj_compute[threadIdx.x] = lrcs_l;
									lrcs_p = k - lrcs_l;
									lrcs_q = seq_idx + k + 1 - lrcs_l;
									lrcs_i = (char)i;
									lrcs_j = (char)j;
								}
							}
							else
								tmp_l = 0;
						}

					}

					idx += blockDim.x;
				}
			}

		}
	}
	__syncthreads();

	j = blockDim.x / 2;
	s_lrcs_tid[threadIdx.x] = threadIdx.x;
	__syncthreads();
	while (j != 0)
	{
		if (threadIdx.x < j && (s_obj_compute[threadIdx.x + j] > s_obj_compute[threadIdx.x]))
		{
			s_obj_compute[threadIdx.x] = s_obj_compute[threadIdx.x + j];
			s_lrcs_tid[threadIdx.x] = s_lrcs_tid[threadIdx.x + j];
		}
		__syncthreads();

		j /= 2;
	}

	if (threadIdx.x == s_lrcs_tid[0])
	{
		ptr_origin_lrcsval[L] = lrcs_l;
		ptr_origin_lrcsval[P] = lrcs_p;
		ptr_origin_lrcsval[Q] = lrcs_q;

		ptr_origin_objval[_MLRCS] = (float)lrcs_l / len_cds;
		ptr_origin_objidx[_MLRCS * 2] = lrcs_i;
		ptr_origin_objidx[_MLRCS * 2 + 1] = lrcs_j;
	}
	__syncthreads();
	/* -------------------- end of initialize -------------------- */



	/* mutate cycle times */
	for (int c = 0; c < cycle; c++)
	{
		/* copy from original solution to target solution */
		num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
		for (i = 0; i < num_partition; i++)
		{
			seq_idx = blockDim.x * i + threadIdx.x;
			if (seq_idx < len_sol)
			{
				ptr_target_sol[seq_idx] = ptr_origin_sol[seq_idx];
			}
		}

		/* select mutatation type */
		if (threadIdx.x == 0) {
			do {
				*mutation_type = (char)(hiprand_uniform(&localState) * 4);
			} while (*mutation_type == 4);
		}
		__syncthreads();


		switch (*mutation_type)
		{
		case 0:			// all random
			num_partition = ((c_len_amino_seq * c_cds_num) % blockDim.x == 0) ? (c_len_amino_seq * c_cds_num) / blockDim.x : (c_len_amino_seq * c_cds_num) / blockDim.x + 1;
			for (i = 0; i < num_partition; i++) {
				idx = blockDim.x * i + threadIdx.x;
				if (idx < c_len_amino_seq * c_cds_num) {
					seq_idx = idx % c_len_amino_seq;

					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_target_sol[idx * CODON_SIZE],
						c_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_target_sol[idx * CODON_SIZE],
						c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);
				}
			}
			break;

		case 1:			// mCAI
			num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
			for (i = 0; i < num_partition; i++) {
				seq_idx = blockDim.x * i + threadIdx.x;
				if (seq_idx < c_len_amino_seq) {
					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mCAI * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mCAI * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, UPPER);
				}
			}
			break;

		case 2:			// mHD
			num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
			for (i = 0; i < num_partition; i++) {
				seq_idx = blockDim.x * i + threadIdx.x;
				if (seq_idx < c_len_amino_seq) {
					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);

					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2 + 1] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
					mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
						&ptr_target_sol[len_cds * ptr_origin_objidx[_mHD * 2 + 1] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);

				}
			}
			break;

		case 3:
			seq_idx = ptr_origin_lrcsval[P] / CODON_SIZE + threadIdx.x;
			while (seq_idx <= (ptr_origin_lrcsval[P] + ptr_origin_lrcsval[L] - 1) / CODON_SIZE)
			{
				pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
					&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
				mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
					&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);

				seq_idx += blockDim.x;
			}

			seq_idx = ptr_origin_lrcsval[Q] / CODON_SIZE + threadIdx.x;
			while (seq_idx <= (ptr_origin_lrcsval[Q] + ptr_origin_lrcsval[L] - 1) / CODON_SIZE)
			{
				pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
					&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2 + 1] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]]);
				mutation(&localState, &c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE],
					&ptr_target_sol[len_cds * ptr_origin_objidx[_MLRCS * 2 + 1] + seq_idx * CODON_SIZE], c_codons_num[s_amino_seq_idx[seq_idx]], pos, c_mprob, RANDOM);

				seq_idx += blockDim.x;
			}

			break;
		}
		__syncthreads();


		/* calculate mCAI */
		num_partition = (c_len_amino_seq % blockDim.x == 0) ? (c_len_amino_seq / blockDim.x) : (c_len_amino_seq / blockDim.x) + 1;
		for (i = 0; i < c_cds_num; i++) {
			s_obj_compute[threadIdx.x] = 1;

			for (j = 0; j < num_partition; j++) {
				seq_idx = blockDim.x * j + threadIdx.x;
				if (seq_idx < c_len_amino_seq) {
					pos = FindNum_C(&c_codons[c_amino_startpos[s_amino_seq_idx[seq_idx]] * CODON_SIZE], &ptr_target_sol[len_cds * i + seq_idx * CODON_SIZE],
						c_codons_num[s_amino_seq_idx[seq_idx]]);
					s_obj_compute[threadIdx.x] *= (float)pow(c_codons_weight[c_amino_startpos[s_amino_seq_idx[seq_idx]] + pos], 1.0 / c_len_amino_seq);
				}
			}
			__syncthreads();

			j = blockDim.x / 2;
			while (j != 0) {
				if (threadIdx.x < j) {
					s_obj_compute[threadIdx.x] *= s_obj_compute[threadIdx.x + j];
				}
				__syncthreads();

				j /= 2;
			}

			if (threadIdx.x == 0) {
				if (i == 0) {
					ptr_target_objval[_mCAI] = s_obj_compute[0];
					ptr_target_objidx[_mCAI * 2] = i;
				}
				else if (s_obj_compute[0] <= ptr_target_objval[_mCAI]) {
					ptr_target_objval[_mCAI] = s_obj_compute[0];
					ptr_target_objidx[_mCAI * 2] = i;
				}
			}
			__syncthreads();

		}

		/* calculate mHD */
		num_partition = (len_cds % blockDim.x == 0) ? (len_cds / blockDim.x) : (len_cds / blockDim.x) + 1;
		for (i = 0; i < c_cds_num; i++) {
			for (j = i + 1; j < c_cds_num; j++) {
				s_obj_compute[threadIdx.x] = 0;

				for (k = 0; k < num_partition; k++) {
					seq_idx = blockDim.x * k + threadIdx.x;

					if (seq_idx < len_cds && (ptr_target_sol[len_cds * i + seq_idx] != ptr_target_sol[len_cds * j + seq_idx])) {
						s_obj_compute[threadIdx.x] += 1;
					}
				}
				__syncthreads();

				k = blockDim.x / 2;
				while (k != 0) {
					if (threadIdx.x < k) {
						s_obj_compute[threadIdx.x] += s_obj_compute[threadIdx.x + k];
					}
					__syncthreads();

					k /= 2;
				}

				if (threadIdx.x == 0) {
					if (i == 0 && j == 1) {
						ptr_target_objval[_mHD] = s_obj_compute[0] / len_cds;
						ptr_target_objidx[_mHD * 2] = i;
						ptr_target_objidx[_mHD * 2 + 1] = j;
					}
					else if (s_obj_compute[0] / len_cds <= ptr_target_objval[_mHD]) {
						ptr_target_objval[_mHD] = s_obj_compute[0] / len_cds;
						ptr_target_objidx[_mHD * 2] = i;
						ptr_target_objidx[_mHD * 2 + 1] = j;
					}
				}
				__syncthreads();

			}
		}

		/* calculate MLRCS */
		s_obj_compute[threadIdx.x] = NOT_FOUND;
		lrcs_l = 0;
		for (i = 0; i < c_cds_num; i++) {
			for (j = i; j < c_cds_num; j++) {
				idx = threadIdx.x;

				if (i == j)
				{
					while (idx < 2 * len_cds + 1)
					{
						if (idx < len_cds + 1) {
							l = idx + 1;
							seq_idx = len_cds - l;

							for (k = 0; k < l; k++) {
								if (k == 0 || (seq_idx == -1))
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + seq_idx + k] == ptr_target_sol[len_cds * j + k - 1]) {
									tmp_l++;
									if (tmp_l >= lrcs_l) {
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = seq_idx + k + 1 - lrcs_l;
										lrcs_q = k - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}
						}
						else {
							l = 2 * len_cds + 1 - idx;
							seq_idx = len_cds - l;

							for (k = 0; k < l; k++) {
								if (k == 0)
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + k - 1] == ptr_target_sol[len_cds * j + seq_idx + k])
								{
									tmp_l++;
									if (tmp_l >= lrcs_l) {
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = k - lrcs_l;
										lrcs_q = seq_idx + k + 1 - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}

						}

						idx += blockDim.x;
					}
				}
				else
				{
					while (idx < 2 * len_cds + 1)
					{
						if (idx < len_cds + 1) {
							l = idx + 1;
							seq_idx = len_cds - l;
							for (k = 0; k < l; k++) {
								if (k == 0)
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + seq_idx + k] == ptr_target_sol[len_cds * j + k - 1]) {
									tmp_l++;
									if (tmp_l >= lrcs_l) {
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = seq_idx + k + 1 - lrcs_l;
										lrcs_q = k - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}
						}
						else {
							l = 2 * len_cds + 1 - idx;
							seq_idx = len_cds - l;

							for (k = 0; k < l; k++) {
								if (k == 0)
									tmp_l = 0;
								else if (ptr_target_sol[len_cds * i + k - 1] == ptr_target_sol[len_cds * j + seq_idx + k])
								{
									tmp_l++;
									if (tmp_l >= lrcs_l) {
										lrcs_l = tmp_l;
										s_obj_compute[threadIdx.x] = lrcs_l;
										lrcs_p = k - lrcs_l;
										lrcs_q = seq_idx + k + 1 - lrcs_l;
										lrcs_i = (char)i;
										lrcs_j = (char)j;
									}
								}
								else
									tmp_l = 0;
							}

						}

						idx += blockDim.x;
					}
				}

			}
		}
		__syncthreads();

		j = blockDim.x / 2;
		s_lrcs_tid[threadIdx.x] = threadIdx.x;
		__syncthreads();
		while (j != 0)
		{
			if (threadIdx.x < j && s_obj_compute[threadIdx.x + j] > s_obj_compute[threadIdx.x])
			{
				s_obj_compute[threadIdx.x] = s_obj_compute[threadIdx.x + j];
				s_lrcs_tid[threadIdx.x] = s_lrcs_tid[threadIdx.x + j];
			}
			__syncthreads();

			j /= 2;
		}

		if (threadIdx.x == s_lrcs_tid[0])
		{
			ptr_target_lrcsval[L] = lrcs_l;
			ptr_target_lrcsval[P] = lrcs_p;
			ptr_target_lrcsval[Q] = lrcs_q;

			ptr_target_objval[_MLRCS] = (float)lrcs_l / len_cds;
			ptr_target_objidx[_MLRCS * 2] = lrcs_i;
			ptr_target_objidx[_MLRCS * 2 + 1] = lrcs_j;
		}
		__syncthreads();


		if (ParetoComparison(ptr_target_objval, ptr_origin_objval))
		{
			if (sol_num == FIRST_SOL)
				sol_num = SECOND_SOL;
			else
				sol_num = FIRST_SOL;
		}

		if (sol_num == FIRST_SOL) {
			ptr_origin_sol = s_sol1;
			ptr_origin_objval = s_sol1_objval;
			ptr_origin_objidx = s_sol1_objidx;
			ptr_origin_lrcsval = s_sol1_lrcsval;
			ptr_target_sol = s_sol2;
			ptr_target_objval = s_sol2_objval;
			ptr_target_objidx = s_sol2_objidx;
			ptr_target_lrcsval = s_sol2_lrcsval;
		}
		else {
			ptr_origin_sol = s_sol2;
			ptr_origin_objval = s_sol2_objval;
			ptr_origin_objidx = s_sol2_objidx;
			ptr_origin_lrcsval = s_sol2_lrcsval;
			ptr_target_sol = s_sol1;
			ptr_target_objval = s_sol1_objval;
			ptr_target_objidx = s_sol1_objidx;
			ptr_target_lrcsval = s_sol1_lrcsval;
		}

	}


	/* copy from shared memory to global memory */
	num_partition = (len_sol % blockDim.x == 0) ? (len_sol / blockDim.x) : (len_sol / blockDim.x) + 1;
	for (i = 0; i < num_partition; i++) {
		idx = blockDim.x * i + threadIdx.x;
		if (idx < len_sol) {
			d_pop[blockIdx.x * len_sol + idx] = ptr_origin_sol[idx];
			d_pop[(gridDim.x + blockIdx.x) * len_sol + idx] = ptr_target_sol[idx];
		}
	}

	if (threadIdx.x == 0)
	{
		d_objval[blockIdx.x * OBJECTIVE_NUM + _mCAI] = ptr_origin_objval[_mCAI];
		d_objval[blockIdx.x * OBJECTIVE_NUM + _mHD] = ptr_origin_objval[_mHD];
		d_objval[blockIdx.x * OBJECTIVE_NUM + _MLRCS] = ptr_origin_objval[_MLRCS];
		d_objval[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM + _mCAI] = ptr_target_objval[_mCAI];
		d_objval[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM + _mHD] = ptr_target_objval[_mHD];
		d_objval[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM + _MLRCS] = ptr_target_objval[_MLRCS];

		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mCAI * 2] = ptr_origin_objidx[_mCAI * 2];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mHD * 2] = ptr_origin_objidx[_mHD * 2];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _mHD * 2 + 1] = ptr_origin_objidx[_mHD * 2 + 1];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _MLRCS * 2] = ptr_origin_objidx[_MLRCS * 2];
		d_objidx[blockIdx.x * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1] = ptr_origin_objidx[_MLRCS * 2 + 1];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _mCAI * 2] = ptr_target_objidx[_mCAI * 2];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _mHD * 2] = ptr_target_objidx[_mHD * 2];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _mHD * 2 + 1] = ptr_target_objidx[_mHD * 2 + 1];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _MLRCS * 2] = ptr_target_objidx[_MLRCS * 2];
		d_objidx[(gridDim.x + blockIdx.x) * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1] = ptr_target_objidx[_MLRCS * 2 + 1];

		d_lrcsval[blockIdx.x * 3 + P] = ptr_origin_lrcsval[P];
		d_lrcsval[blockIdx.x * 3 + Q] = ptr_origin_lrcsval[Q];
		d_lrcsval[blockIdx.x * 3 + L] = ptr_origin_lrcsval[L];
		d_lrcsval[(gridDim.x + blockIdx.x) * 3 + P] = ptr_target_lrcsval[P];
		d_lrcsval[(gridDim.x + blockIdx.x) * 3 + Q] = ptr_target_lrcsval[Q];
		d_lrcsval[(gridDim.x + blockIdx.x) * 3 + L] = ptr_target_lrcsval[L];
	}

	state[id] = localState;						// update state of random number generator

	return;
}


int main()
{
	srand((unsigned int)time(NULL));

	/* To get information of Deivce */
	int dev = 0;							// number of device (GPU)
	int maxSharedMemPerBlock;
	int maxSharedMemPerProcessor;
	int totalConstantMem;
	int maxRegisterPerProcessor;
	int maxRegisterPerBlock;
	int totalMultiProcessor;
	hipDeviceProp_t deviceProp;

	CHECK_CUDA(hipGetDeviceProperties(&deviceProp, dev))
		CHECK_CUDA(hipDeviceGetAttribute(&maxSharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, dev))
		CHECK_CUDA(hipDeviceGetAttribute(&maxSharedMemPerProcessor, hipDeviceAttributeMaxRegistersPerMultiprocessor, dev))
		CHECK_CUDA(hipDeviceGetAttribute(&totalConstantMem, hipDeviceAttributeTotalConstantMemory, dev))
		CHECK_CUDA(hipDeviceGetAttribute(&maxRegisterPerProcessor, hipDeviceAttributeMaxRegistersPerMultiprocessor, dev))
		CHECK_CUDA(hipDeviceGetAttribute(&maxRegisterPerBlock, hipDeviceAttributeMaxRegistersPerBlock, dev))
		CHECK_CUDA(hipDeviceGetAttribute(&totalMultiProcessor, hipDeviceAttributeMultiprocessorCount, dev))

		printf("Device #%d:\n", dev);
	printf("Name: %s\n", deviceProp.name);
	printf("Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("Clock rate: %d MHz\n", deviceProp.clockRate / 1000);
	printf("Global memory size: %lu MB\n", deviceProp.totalGlobalMem / (1024 * 1024));
	printf("Max thread dimensions: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("Max grid dimensions: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("Total constant memory: %d bytes\n", totalConstantMem);
	printf("Max threads per SM: %d\n", deviceProp.maxThreadsPerMultiProcessor);
	printf("Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
	printf("Maximum shared memory per SM: %d bytes\n", maxSharedMemPerProcessor);
	printf("Maximum shared memory per block: %d bytes\n", maxSharedMemPerBlock);
	printf("Maximum number of registers per SM: %d\n", maxRegisterPerProcessor);
	printf("Maximum number of registers per block: %d\n", maxRegisterPerBlock);
	printf("Total number of SM in device: %d\n", totalMultiProcessor);
	printf("\n");

	char input_file[32];
	char* amino_seq;						// store amino sequences from input file
	char* h_amino_seq_idx;					// notify index of amino abbreviation array corresponding input amino sequences
	char* h_amino_startpos;					// notify position of according amino abbreviation index
	char* h_pop;							// store population (a set of solutions)
	float* h_objval;						// store objective values of population (solution 1, solution 2 .... solution n)
	char* h_objidx;
	int* h_lrcsval;
	int len_amino_seq, len_cds, len_sol;
	int pop_size;
	int cycle;
	int cds_num;							// size of solution equal to number of CDSs(codon sequences) in a solution
	float mprob;							// mutation probability
	float min_dist;


	char tmp;
	int i, j, k;
	int x;
	int idx;
	char buf[256];
	FILE* fp;


	int numBlocks;
	int threadsPerBlock;

	char* d_amino_seq_idx;
	char* d_pop;
	float* d_objval;
	char* d_objidx;
	int* d_lrcsval;
	hiprandStateXORWOW* genState;

	/* for time and mcai section cehck */
	hipEvent_t d_start, d_end;
	float kernel_time;
	hipEventCreate(&d_start);
	hipEventCreate(&d_end);



	/* ---------------------------------------- preprocessing ---------------------------------------- */
	/* input parameter values */
	printf("input file name : "); scanf("%s", input_file);
	printf("input number of cycle : "); scanf("%d", &cycle);					// if number of cycle is zero we can check initial population
	if (cycle < 0) {
		printf("input max cycle value >= 0\n");
		return EXIT_FAILURE;
	}
	printf("input number of solution : "); scanf("%d", &pop_size);
	if (pop_size <= 0) {
		printf("input number of solution > 0\n");
		return EXIT_FAILURE;
	}
	printf("input number of CDSs in a solution : "); scanf("%d", &cds_num);
	if (cds_num <= 1) {
		printf("input number of CDSs > 1\n");
		return EXIT_FAILURE;
	}
	printf("input mutation probability (0 ~ 1 value) : "); scanf("%f", &mprob);
	if (mprob < 0 || mprob > 1) {
		printf("input mutation probability (0 ~ 1 value) : \n");
		return EXIT_FAILURE;
	}
	printf("input thread per block x value --> number of thread  warp size (32) * x : "); scanf("%d", &x);


	/* read input file (fasta format) */
	fp = fopen(input_file, "r");
	if (fp == NULL) {
		printf("Line : %d Opening input file is failed", __LINE__);
		return EXIT_FAILURE;
	}

	fseek(fp, 0, SEEK_END);
	len_amino_seq = ftell(fp);
	fseek(fp, 0, SEEK_SET);
	fgets(buf, 256, fp);
	len_amino_seq -= ftell(fp);

	amino_seq = (char*)malloc(sizeof(char) * len_amino_seq);

	idx = 0;
	while (!feof(fp)) {
		tmp = fgetc(fp);
		if (tmp != '\n')
			amino_seq[idx++] = tmp;
	}
	amino_seq[idx] = NULL;
	len_amino_seq = idx - 1;
	len_cds = len_amino_seq * CODON_SIZE;
	len_sol = len_cds * cds_num;

	fclose(fp);
	/* end file process */

	h_amino_seq_idx = (char*)malloc(sizeof(char) * len_amino_seq);
	for (i = 0; i < len_amino_seq; i++) {
		idx = FindAminoIndex(amino_seq[i]);
		if (idx == NOT_FOUND) {
			printf("FindAminoIndex function is failed... \n");
			return EXIT_FAILURE;
		}
		h_amino_seq_idx[i] = idx;
	}

	h_amino_startpos = (char*)malloc(sizeof(char) * 20);
	h_amino_startpos[0] = 0;
	for (i = 1; i < 20; i++) {
		h_amino_startpos[i] = h_amino_startpos[i - 1] + Codons_num[i - 1];
	}
	/* ---------------------------------------- end of preprocessing ---------------------------------------- */


	numBlocks = pop_size;
	threadsPerBlock = WARP_SIZE * x;

	/* host memory allocation */
	h_pop = (char*)malloc(sizeof(char) * pop_size * len_sol * 2);
	h_objval = (float*)malloc(sizeof(float) * pop_size * OBJECTIVE_NUM * 2);
	h_objidx = (char*)malloc(sizeof(char) * pop_size * OBJECTIVE_NUM * 2 * 2);
	h_lrcsval = (int*)malloc(sizeof(int) * pop_size * 3 * 2);



	/* device memory allocation */
	CHECK_CUDA(hipMalloc((void**)&genState, sizeof(hiprandStateXORWOW) * numBlocks * threadsPerBlock))
		CHECK_CUDA(hipMalloc((void**)&d_amino_seq_idx, sizeof(char) * len_amino_seq))
		CHECK_CUDA(hipMalloc((void**)&d_pop, sizeof(char) * numBlocks * len_sol * 2))
		CHECK_CUDA(hipMalloc((void**)&d_objval, sizeof(float) * numBlocks * OBJECTIVE_NUM * 2))
		CHECK_CUDA(hipMalloc((void**)&d_objidx, sizeof(char) * numBlocks * OBJECTIVE_NUM * 2 * 2))
		CHECK_CUDA(hipMalloc((void**)&d_lrcsval, sizeof(int) * numBlocks * 3 * 2))


		/* memory copy host to device */
		CHECK_CUDA(hipMemcpy(d_amino_seq_idx, h_amino_seq_idx, sizeof(char) * len_amino_seq, hipMemcpyHostToDevice))
		CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_codons_weight), Codons_weight, sizeof(Codons_weight)))
		CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_amino_startpos), h_amino_startpos, sizeof(char) * 20))
		CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_codons), Codons, sizeof(Codons)))
		CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_codons_num), Codons_num, sizeof(Codons_num)))
		CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_len_amino_seq), &len_amino_seq, sizeof(int)))
		CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_cds_num), &cds_num, sizeof(int)))
		CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(c_mprob), &mprob, sizeof(float)))

		/* ------------------------------------------------ kerenl call ----------------------------------------------- */
		setup_kernel << <numBlocks, threadsPerBlock >> > (genState, rand());

	hipEventRecord(d_start);
	mainKernel << <numBlocks, threadsPerBlock,
		sizeof(int)* (threadsPerBlock + 3 * 2) + sizeof(float) * (threadsPerBlock + OBJECTIVE_NUM * 2) + sizeof(char) * (len_sol * 2 + len_amino_seq + OBJECTIVE_NUM * 2 * 2 + 1) >> >
		(genState, d_amino_seq_idx, d_pop, d_objval, d_objidx, d_lrcsval, cycle);
	hipEventRecord(d_end);
	hipEventSynchronize(d_end);
	hipEventElapsedTime(&kernel_time, d_start, d_end);
	printf("\nGPU kerenl cycle time : %f second\n\n", kernel_time / 1000.f);
	printf("using shared memory size : %lu\n", sizeof(int) * (threadsPerBlock + 3 * 2) + sizeof(float) * (threadsPerBlock + OBJECTIVE_NUM * 2) + sizeof(char) * (len_sol * 2 + len_amino_seq + OBJECTIVE_NUM * 2 * 2 + 1));
	printf("using contant memory size : %lu\n\n", sizeof(Codons_weight) + sizeof(char) * 20 + sizeof(Codons) + sizeof(Codons_num) + sizeof(int) * 2 + sizeof(float));


	/* memory copy device to host */
	CHECK_CUDA(hipMemcpy(h_pop, d_pop, sizeof(char) * numBlocks * len_sol * 2, hipMemcpyDeviceToHost))
		CHECK_CUDA(hipMemcpy(h_objval, d_objval, sizeof(float) * numBlocks * OBJECTIVE_NUM * 2, hipMemcpyDeviceToHost))
		CHECK_CUDA(hipMemcpy(h_objidx, d_objidx, sizeof(char) * numBlocks * OBJECTIVE_NUM * 2 * 2, hipMemcpyDeviceToHost))
		CHECK_CUDA(hipMemcpy(h_lrcsval, d_lrcsval, sizeof(int) * numBlocks * 3 * 2, hipMemcpyDeviceToHost))


		// for compute hypervolume & minimum distance out process
		for (i = 0; i < pop_size * 2; i++)
		{
			h_objval[i * OBJECTIVE_NUM + _mHD] /= 0.4;
		}



		// print minimum distance to ideal point
		min_dist = MinEuclid(h_objval, pop_size);
	printf("\nminimum distance to the ideal point : %f\n", min_dist);


	/* print solution */
	//for (i = 0; i < pop_size * 2; i++)
	//{
	//	printf("%d solution\n", i + 1);
	//	for (j = 0; j < cds_num; j++) {
	//		printf("%d cds : ", j + 1);
	//		for (k = 0; k < len_cds; k++) {
	//			printf("%c", h_pop[len_sol * i + len_cds * j + k]);
	//		}
	//		printf("\n");
	//	}
	//	printf("\n");
	//}


	/* print objective value */
	//for (i = 0; i < pop_size * 2; i++)
	//{
	//	printf("%d solution\n", i + 1);
	//	printf("mCAI : %f mHD : %f MLRCS : %f\n", h_objval[i * OBJECTIVE_NUM + _mCAI], h_objval[i * OBJECTIVE_NUM + _mHD], h_objval[i * OBJECTIVE_NUM + _MLRCS]);
	//	printf("mCAI idx : %d mHD idx : %d %d MLRCS idx : %d %d\n", h_objidx[i * OBJECTIVE_NUM * 2 + _mCAI * 2],
	//		h_objidx[i * OBJECTIVE_NUM * 2 + _mHD * 2], h_objidx[i * OBJECTIVE_NUM * 2 + _mHD * 2 + 1],
	//		h_objidx[i * OBJECTIVE_NUM * 2 + _MLRCS * 2], h_objidx[i * OBJECTIVE_NUM * 2 + _MLRCS * 2 + 1]);
	//	printf("P : %d Q : %d L : %d\n", h_lrcsval[i * 3 + P], h_lrcsval[i * 3 + Q], h_lrcsval[i * 3 + L]);
	//}


	fp = fopen("test.txt", "w");
	/* for computing hypervolume write file */
	for (i = 0; i < pop_size * 2; i++)
	{
		fprintf(fp, "%f %f %f\n", -h_objval[i * OBJECTIVE_NUM + _mCAI], -h_objval[i * OBJECTIVE_NUM + _mHD], h_objval[i * OBJECTIVE_NUM + _MLRCS]);
	}
	fclose(fp);



	/* free deivce memory */
	CHECK_CUDA(hipFree(genState))
		CHECK_CUDA(hipFree(d_amino_seq_idx))
		CHECK_CUDA(hipFree(d_pop))
		CHECK_CUDA(hipFree(d_objval))
		CHECK_CUDA(hipFree(d_objidx))
		CHECK_CUDA(hipFree(d_lrcsval))
		CHECK_CUDA(hipEventDestroy(d_start))
		CHECK_CUDA(hipEventDestroy(d_end))

		/* free host memory */
		free(amino_seq);
	free(h_amino_seq_idx);
	free(h_amino_startpos);
	free(h_pop);
	free(h_objval);
	free(h_objidx);
	free(h_lrcsval);


	return EXIT_SUCCESS;
}